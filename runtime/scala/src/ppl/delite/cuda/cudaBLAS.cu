#include <stdio.h>
#include "hip/hip_runtime.h"
#include "ppl_delite_cuda_DeliteCudaDriver.h"

/* Global variables for primitive type classes and get methods.
   Initialized in initIDs() function
*/
jclass CLS_Integer;
jclass CLS_Long;
jclass CLS_Float;
jclass CLS_Double;
jmethodID MID_intValue;
jmethodID MID_longValue;
jmethodID MID_floatValue;
jmethodID MID_doubleValue;
jmethodID MID_getArg;
jmethodID MID_getArgSize;

JNIEXPORT void JNICALL Java_ppl_delite_cuda_DeliteCudaDriver_initIDs(JNIEnv *env, jobject obj)
{
	// Get local reference to primitive type classes
	jclass local_CLS_Integer = env->FindClass("java/lang/Integer");
	jclass local_CLS_Long = env->FindClass("java/lang/Long");
	jclass local_CLS_Float = env->FindClass("java/lang/Float");
	jclass local_CLS_Double = env->FindClass("java/lang/Double");

	// Save local reference to global references for later uses 
	CLS_Integer = (jclass)(env->NewGlobalRef(local_CLS_Integer));
	CLS_Long = (jclass)(env->NewGlobalRef(local_CLS_Long));
	CLS_Float = (jclass)(env->NewGlobalRef(local_CLS_Float));
	CLS_Double = (jclass)(env->NewGlobalRef(local_CLS_Double));

	// Get method IDs for get methods of primitive type classes
	MID_intValue = env->GetMethodID(CLS_Integer, "intValue", "()I");
	MID_longValue = env->GetMethodID(CLS_Long, "longValue", "()J");
	MID_floatValue = env->GetMethodID(CLS_Float, "floatValue", "()F");
	MID_doubleValue = env->GetMethodID(CLS_Double, "doubleValue", "()D");
	
	// Get class/method ID for argument list queue
	jclass local_CLS_Queue = env->FindClass("java/util/concurrent/ArrayBlockingQueue");
	MID_getArg = env->GetMethodID(local_CLS_Queue, "poll", "()Ljava/lang/Object;");
	MID_getArgSize = env->GetMethodID(local_CLS_Queue, "size", "()I");
}

JNIEXPORT void JNICALL Java_ppl_delite_cuda_DeliteCudaDriver_cudaAsyncLaunch1D(JNIEnv *env, jobject obj, jlong func_id, jobject argsList, jint length1D, jlong stream)
{
	int i = 0;
	int offset = 0;
	hipFunction_t func = (hipFunction_t)func_id;
	jobject arg;
	int argsSize = env->CallIntMethod(argsList, MID_getArgSize);

	while(i < argsSize) {
		arg = env->CallObjectMethod(argsList, MID_getArg);
		if(env->IsInstanceOf(arg, CLS_Integer)) {
			int value = env->CallIntMethod(arg, MID_intValue);
			offset = (offset + __alignof(value) - 1) & ~(__alignof(value) - 1);
			if(cuParamSeti(func, offset,  value) != hipSuccess) {
				printf("ERROR: cuParamSeti (Async1D)\n");
				return;
			}
			offset += sizeof(value);
			//printf("Integer : %d\n", value);
		}
		else if(env->IsInstanceOf(arg, CLS_Long)) {
			long value = env->CallLongMethod(arg, MID_longValue);
			offset = (offset + __alignof(value) - 1) & ~(__alignof(value) - 1);
			if(cuParamSetv(func, offset, &value, sizeof(value)) != hipSuccess) {
				printf("ERROR: cuParamSetv (Async1D)\n");
				return;
			}
			offset += sizeof(value);
			//printf("Long : %ld\n", value);
		}
		else if(env->IsInstanceOf(arg, CLS_Float)) {
		    float value = env->CallFloatMethod(arg, MID_floatValue);
			offset = (offset + __alignof(value) - 1) & ~(__alignof(value) - 1);
			if(cuParamSetv(func, offset, &value, sizeof(value)) != hipSuccess) {
				printf("ERROR: cuParamSetv (Async1D)\n");
				return;
			}
			offset += sizeof(value);
			//printf("Float : %f\n", env->CallFloatMethod(arg, MID_floatValue));
		}
		else if(env->IsInstanceOf(arg, CLS_Double)) {
		    double value = env->CallDoubleMethod(arg, MID_doubleValue);
			offset = (offset + __alignof(value) - 1) & ~(__alignof(value) - 1);
			if(cuParamSetv(func, offset, &value, sizeof(value)) != hipSuccess) {
				printf("ERROR: cuParamSetv (Async1D)\n");
				return;
			}
			offset += sizeof(value);
			//printf("Double : %f\n", env->CallDoubleMethod(arg, MID_doubleValue));
		}
		else {
			printf("Not Supported Type to pass to GPU.\n");
		}
		i += 1;
	}

	if(cuParamSetSize(func, offset) != hipSuccess) {
		printf("ERROR: cuParamSetSize (Async1D)\n");
		return;
	}
	
	if(cuFuncSetBlockShape(func, 512, 1, 1) != hipSuccess) {
		printf("ERROR: cuFuncSetBlockShape (Async1D)\n");
		return;
	}
	if(cuLaunchGridAsync(func, 1+(length1D-1)/512, 1, (hipStream_t)stream) != hipSuccess) {
		printf("ERROR: cuLaunchGrid (Async1D)\n");
		return;
	}
}

JNIEXPORT void JNICALL Java_ppl_delite_cuda_DeliteCudaDriver_cudaAsyncLaunch2D(JNIEnv *env, jobject obj, jlong func_id, jobject argsList, jint length1D, jint length2D, jlong stream)
{
	int i = 0;
	int offset = 0;
	hipFunction_t func = (hipFunction_t)func_id;
	jobject arg;
    int argsSize = env->CallIntMethod(argsList, MID_getArgSize);
    
	while(i < argsSize) {
		arg = env->CallObjectMethod(argsList, MID_getArg);
		if(env->IsInstanceOf(arg, CLS_Integer)) {
			int value = env->CallIntMethod(arg, MID_intValue);
			offset = (offset + __alignof(value) - 1) & ~(__alignof(value) - 1);
			if(cuParamSeti(func, offset,  value) != hipSuccess) {
				printf("ERROR: cuParamSeti (Async2D)\n");
				return;
			}
			offset += sizeof(value);
			//printf("Integer : %d\n", value);
		}
		else if(env->IsInstanceOf(arg, CLS_Long)) {
			long value = env->CallLongMethod(arg, MID_longValue);
			offset = (offset + __alignof(value) - 1) & ~(__alignof(value) - 1);
			if(cuParamSetv(func, offset, &value, sizeof(value)) != hipSuccess) {
				printf("ERROR: cuParamSetv (Async2D)\n");
				return;
			}
			offset += sizeof(value);
			//printf("Long : %ld\n", value);
		}
		else if(env->IsInstanceOf(arg, CLS_Float)) {
		    float value = env->CallFloatMethod(arg, MID_floatValue);
			offset = (offset + __alignof(value) - 1) & ~(__alignof(value) - 1);
			if(cuParamSetv(func, offset, &value, sizeof(value)) != hipSuccess) {
				printf("ERROR: cuParamSetv (Async2D)\n");
				return;
			}
			offset += sizeof(value);
			//printf("Float : %f\n", env->CallFloatMethod(arg, MID_floatValue));
		}
		else if(env->IsInstanceOf(arg, CLS_Double)) {
		    double value = env->CallDoubleMethod(arg, MID_doubleValue);
			offset = (offset + __alignof(value) - 1) & ~(__alignof(value) - 1);
			if(cuParamSetv(func, offset, &value, sizeof(value)) != hipSuccess) {
				printf("ERROR: cuParamSetv (Async2D)\n");
				return;
			}
			offset += sizeof(value);
			//printf("Double : %f\n", env->CallDoubleMethod(arg, MID_doubleValue));
		}
		else {
			printf("Not Supported Type to pass to GPU.\n");
		}
		i += 1;
	}

	if(cuParamSetSize(func, offset) != hipSuccess) {
		printf("ERROR: cuParamSetSize (Async2D)\n");
		return;
	}
	
	if(cuFuncSetBlockShape(func, 32, 16, 1) != hipSuccess) {
		printf("ERROR: cuFuncSetBlockShape (Async2D)\n");
		return;
	}
	if(cuLaunchGridAsync(func, 1+(length1D-1)/32, 1+(length2D-1)/16, (hipStream_t)stream) != hipSuccess) {
		printf("ERROR: cuLaunchGrid (Async2D)\n");
		return;
	}
}

JNIEXPORT void JNICALL Java_ppl_delite_cuda_DeliteCudaDriver_cudaAsyncLaunch1DSpec(JNIEnv *env, jobject obj, jlong func_id, jobject argsList, jint length1D, jint blockDim1D, jlong stream)
{
	int i = 0;
	int offset = 0;
	hipFunction_t func = (hipFunction_t)func_id;
	jobject arg;
	int argsSize = env->CallIntMethod(argsList, MID_getArgSize);

	while(i < argsSize) {
		arg = env->CallObjectMethod(argsList, MID_getArg);
		if(env->IsInstanceOf(arg, CLS_Integer)) {
			int value = env->CallIntMethod(arg, MID_intValue);
			offset = (offset + __alignof(value) - 1) & ~(__alignof(value) - 1);
			if(cuParamSeti(func, offset,  value) != hipSuccess) {
				printf("ERROR: cuParamSeti (Async1D)\n");
				return;
			}
			offset += sizeof(value);
			//printf("Integer : %d\n", value);
		}
		else if(env->IsInstanceOf(arg, CLS_Long)) {
			long value = env->CallLongMethod(arg, MID_longValue);
			offset = (offset + __alignof(value) - 1) & ~(__alignof(value) - 1);
			if(cuParamSetv(func, offset, &value, sizeof(value)) != hipSuccess) {
				printf("ERROR: cuParamSetv (Async1D)\n");
				return;
			}
			offset += sizeof(value);
			//printf("Long : %ld\n", value);
		}
		else if(env->IsInstanceOf(arg, CLS_Float)) {
		    float value = env->CallFloatMethod(arg, MID_floatValue);
			offset = (offset + __alignof(value) - 1) & ~(__alignof(value) - 1);
			if(cuParamSetv(func, offset, &value, sizeof(value)) != hipSuccess) {
				printf("ERROR: cuParamSetv (Async1D)\n");
				return;
			}
			offset += sizeof(value);
			//printf("Float : %f\n", env->CallFloatMethod(arg, MID_floatValue));
		}
		else if(env->IsInstanceOf(arg, CLS_Double)) {
		    double value = env->CallDoubleMethod(arg, MID_doubleValue);
			offset = (offset + __alignof(value) - 1) & ~(__alignof(value) - 1);
			if(cuParamSetv(func, offset, &value, sizeof(value)) != hipSuccess) {
				printf("ERROR: cuParamSetv (Async1D)\n");
				return;
			}
			offset += sizeof(value);
			//printf("Double : %f\n", env->CallDoubleMethod(arg, MID_doubleValue));
		}
		else {
			printf("Not Supported Type to pass to GPU.\n");
		}
		i += 1;
	}

	if(cuParamSetSize(func, offset) != hipSuccess) {
		printf("ERROR: cuParamSetSize (Async1D)\n");
		return;
	}

	if(cuFuncSetBlockShape(func, blockDim1D, 1, 1) != hipSuccess) {
		printf("ERROR: cuFuncSetBlockShape (Async1D)\n");
		return;
	}
	if(cuLaunchGridAsync(func, 1+(length1D-1)/blockDim1D, 1, (hipStream_t)stream) != hipSuccess) {
		printf("ERROR: cuLaunchGrid (Async1D)\n");
		return;
	}
}

JNIEXPORT void JNICALL Java_ppl_delite_cuda_DeliteCudaDriver_cudaAsyncLaunch2DSpec(JNIEnv *env, jobject obj, jlong func_id, jobject argsList, jint length1D, jint length2D, jint blockDim1D, jint blockDim2D, jlong stream)
{
	int i = 0;
	int offset = 0;
	hipFunction_t func = (hipFunction_t)func_id;
	jobject arg;
    int argsSize = env->CallIntMethod(argsList, MID_getArgSize);

	while(i < argsSize) {
		arg = env->CallObjectMethod(argsList, MID_getArg);
		if(env->IsInstanceOf(arg, CLS_Integer)) {
			int value = env->CallIntMethod(arg, MID_intValue);
			offset = (offset + __alignof(value) - 1) & ~(__alignof(value) - 1);
			if(cuParamSeti(func, offset,  value) != hipSuccess) {
				printf("ERROR: cuParamSeti (Async2D)\n");
				return;
			}
			offset += sizeof(value);
			//printf("Integer : %d\n", value);
		}
		else if(env->IsInstanceOf(arg, CLS_Long)) {
			long value = env->CallLongMethod(arg, MID_longValue);
			offset = (offset + __alignof(value) - 1) & ~(__alignof(value) - 1);
			if(cuParamSetv(func, offset, &value, sizeof(value)) != hipSuccess) {
				printf("ERROR: cuParamSetv (Async2D)\n");
				return;
			}
			offset += sizeof(value);
			//printf("Long : %ld\n", value);
		}
		else if(env->IsInstanceOf(arg, CLS_Float)) {
		    float value = env->CallFloatMethod(arg, MID_floatValue);
			offset = (offset + __alignof(value) - 1) & ~(__alignof(value) - 1);
			if(cuParamSetv(func, offset, &value, sizeof(value)) != hipSuccess) {
				printf("ERROR: cuParamSetv (Async2D)\n");
				return;
			}
			offset += sizeof(value);
			//printf("Float : %f\n", env->CallFloatMethod(arg, MID_floatValue));
		}
		else if(env->IsInstanceOf(arg, CLS_Double)) {
		    double value = env->CallDoubleMethod(arg, MID_doubleValue);
			offset = (offset + __alignof(value) - 1) & ~(__alignof(value) - 1);
			if(cuParamSetv(func, offset, &value, sizeof(value)) != hipSuccess) {
				printf("ERROR: cuParamSetv (Async2D)\n");
				return;
			}
			offset += sizeof(value);
			//printf("Double : %f\n", env->CallDoubleMethod(arg, MID_doubleValue));
		}
		else {
			printf("Not Supported Type to pass to GPU.\n");
		}
		i += 1;
	}

	if(cuParamSetSize(func, offset) != hipSuccess) {
		printf("ERROR: cuParamSetSize (Async2D)\n");
		return;
	}

	if(cuFuncSetBlockShape(func, blockDim1D, blockDim2D, 1) != hipSuccess) {
		printf("ERROR: cuFuncSetBlockShape (Async2D)\n");
		return;
	}
	if(cuLaunchGridAsync(func, 1+(length1D-1)/blockDim1D, 1+(length2D-1)/blockDim2D, (hipStream_t)stream) != hipSuccess) {
		printf("ERROR: cuLaunchGrid (Async2D)\n");
		return;
	}
}

JNIEXPORT void JNICALL Java_ppl_delite_cuda_DeliteCudaDriver_cudaAsync3D3I(JNIEnv *env, jobject obj, jlong func_id, jlong a_ptr, jlong b_ptr, jlong c_ptr, jint widthA, jint widthB, jint heightA, jlong stream)
{
	int offset = 0;
	void* ptr;
	hipFunction_t func = (hipFunction_t)func_id;

	ptr = (void*)(size_t)a_ptr;
	offset = (offset + __alignof(ptr) - 1) & ~(__alignof(ptr) - 1);
	if(cuParamSetv(func, offset, &ptr, sizeof(ptr)) != hipSuccess)
	{
		printf("ERROR: cuParamSetv (cudaAsync3D3I)\n");
		return;
	}
	offset += sizeof(ptr);

	ptr = (void*)(size_t)b_ptr;
	offset = (offset + __alignof(ptr) - 1) & ~(__alignof(ptr) - 1);
	if(cuParamSetv(func, offset, &ptr, sizeof(ptr)) != hipSuccess)
	{
		printf("ERROR: cuParamSetv (cudaAsync3D3I)\n");
		return;
	}
	offset += sizeof(ptr);

	ptr = (void*)(size_t)c_ptr;
	offset = (offset + __alignof(ptr) - 1) & ~(__alignof(ptr) - 1);
	if(cuParamSetv(func, offset, &ptr, sizeof(ptr)) != hipSuccess)
	{
		printf("ERROR: cuParamSetv (cudaAsync3D3I)\n");
		return;
	}
	offset += sizeof(ptr);

	offset = (offset + __alignof(widthA) - 1) & ~(__alignof(widthA) - 1);
	if(cuParamSeti(func, offset, widthA) != hipSuccess)
	{
		printf("ERROR: cuParamSetv (cudaAsync3D3I)\n");
		return;
	}
	offset += sizeof(widthA);

	offset = (offset + __alignof(widthB) - 1) & ~(__alignof(widthB) - 1);
	if(cuParamSeti(func, offset,  widthB) != hipSuccess)
	{
		printf("ERROR: cuParamSetv (cudaAsync3D3I)\n");
		return;
	}
	offset += sizeof(widthB);

	offset = (offset + __alignof(heightA) - 1) & ~(__alignof(heightA) - 1);
	if(cuParamSeti(func, offset, heightA) != hipSuccess)
	{
		printf("ERROR: cuParamSetv (cudaAsync3D3I)\n");
		return;
	}
	offset += sizeof(heightA);


	if(cuParamSetSize(func, offset) != hipSuccess)
	{
		printf("ERROR: cuParamSetSize (cudaAsync3D3I)\n");
		return;
	}

    //TODO : Need to receive dimension info as inputs
    if(widthB == 0) {
        if(cuFuncSetBlockShape(func, 16, 16, 1) != hipSuccess)
        {
            printf("ERROR: cuFuncSetBlockShape\n");
            return;
        }
        if(cuLaunchGridAsync(func, 1+(widthA-1)/16, 1+(heightA-1)/16, (hipStream_t)stream) != hipSuccess)
        {
            printf("ERROR: cuLaunchGrid\n");
            return;
        }
    }
    else {
        if(cuFuncSetBlockShape(func, 16, 4, 1) != hipSuccess)
        //if(cuFuncSetBlockShape(func, 16, 16, 1) != hipSuccess)
        {
            printf("ERROR: cuFuncSetBlockShape\n");
            return;
        }
        //if(cuLaunchGridAsync(func, 1+(widthB-1)/16, 1+(heightA-1)/16, (hipStream_t)stream) != hipSuccess)
        if(cuLaunchGridAsync(func, (widthB+63)/64, (heightA+15)/16, (hipStream_t)stream) != hipSuccess)
        {
            printf("ERROR: cuLaunchGrid\n");
            return;
        }
    }
}


/*
JNIEXPORT void JNICALL Java_ppl_delite_cuda_DeliteCudaDriver_cudaBLAS1D(JNIEnv *env, jobject obj, jlong func_id, jlong a_ptr, jlong b_ptr, jlong c_ptr, jint count)
{
	int offset = 0;
	void* ptr;
	hipFunction_t func = (hipFunction_t)func_id;

	ptr = (void*)(size_t)a_ptr;
	offset = (offset + __alignof(ptr) - 1) & ~(__alignof(ptr) - 1);
	if(cuParamSetv(func, offset, &ptr, sizeof(ptr)) != hipSuccess)
	{
		printf("ERROR: cuParamSetv (cudaBLAS1D) \n");
		return;
	}
	offset += sizeof(ptr);

	ptr = (void*)(size_t)b_ptr;
	offset = (offset + __alignof(ptr) - 1) & ~(__alignof(ptr) - 1); 
	if(cuParamSetv(func, offset, &ptr, sizeof(ptr)) != hipSuccess)
	{
		printf("ERROR: cuParamSetv (cudaBLAS1D) \n");
		return;
	}
	offset += sizeof(ptr);

	ptr = (void*)(size_t)c_ptr;
	offset = (offset + __alignof(ptr) - 1) & ~(__alignof(ptr) - 1);
	if(cuParamSetv(func, offset, &ptr, sizeof(ptr)) != hipSuccess)
	{
		printf("ERROR: cuParamSetv (cudaBLAS1D) \n");
		return;
	}
	offset += sizeof(ptr);

	offset = (offset + __alignof(count) - 1) & ~(__alignof(count) - 1);
	if(cuParamSeti(func, offset, count) != hipSuccess)
	{
		printf("ERROR: cuParamSetv (cudaBLAS1D) \n");
		return;
	}
	offset += sizeof(count);

	if(cuParamSetSize(func, offset) != hipSuccess)
	{
		printf("ERROR: cuParamSetSize (cudaBLAS1D) \n");
		return;
	}
	
	//TODO : Need to receive dimension info as inputs
	if(cuFuncSetBlockShape(func, 512, 1, 1) != hipSuccess)
	{
		printf("ERROR: cuFuncSetBlockShape (cudaBLAS1D)\n");
		return;
	}

	if(cuLaunchGrid(func, 1 + (count-1)/512, 1) != hipSuccess)
	{
		printf("ERROR: cuLaunchGrid (cudaBLAS1D)\n");
		return;
	}

	if(hipCtxSynchronize() != hipSuccess)
    {
        printf("ERROR: cudaContextSync (cudaBLAS1D)\n");
        return;
    }
}

JNIEXPORT void JNICALL Java_ppl_delite_cuda_DeliteCudaDriver_cudaBLAS2D(JNIEnv *env, jobject obj, jlong func_id, jlong a_ptr, jlong b_ptr, jlong c_ptr, jint count)
{
	int offset = 0;
	void* ptr;
	hipFunction_t func = (hipFunction_t)func_id;

	ptr = (void*)(size_t)a_ptr;
	offset = (offset + __alignof(ptr) - 1) & ~(__alignof(ptr) - 1);
	if(cuParamSetv(func, offset, &ptr, sizeof(ptr)) != hipSuccess)
	{
		printf("ERROR: cuParamSetv (cudaBLAS2D) \n");
		return;
	}
	offset += sizeof(ptr);

	ptr = (void*)(size_t)b_ptr;
	offset = (offset + __alignof(ptr) - 1) & ~(__alignof(ptr) - 1); 
	if(cuParamSetv(func, offset, &ptr, sizeof(ptr)) != hipSuccess)
	{
		printf("ERROR: cuParamSetv (cudaBLAS2D) \n");
		return;
	}
	offset += sizeof(ptr);

	ptr = (void*)(size_t)c_ptr;
	offset = (offset + __alignof(ptr) - 1) & ~(__alignof(ptr) - 1);
	if(cuParamSetv(func, offset, &ptr, sizeof(ptr)) != hipSuccess)
	{
		printf("ERROR: cuParamSetv (cudaBLAS2D) \n");
		return;
	}
	offset += sizeof(ptr);

	offset = (offset + __alignof(count) - 1) & ~(__alignof(count) - 1);
	if(cuParamSeti(func, offset, count) != hipSuccess)
	{
		printf("ERROR: cuParamSetv (cudaBLAS2D) \n");
		return;
	}
	offset += sizeof(count);

	if(cuParamSetSize(func, offset) != hipSuccess)
	{
		printf("ERROR: cuParamSetSize (cudaBLAS2D) \n");
		return;
	}
	
	//TODO : Need to receive dimension info as inputs
	if(cuFuncSetBlockShape(func, 16, 1, 1) != hipSuccess)
	{
		printf("ERROR: cuFuncSetBlockShape (cudaBLAS2D)\n");
		return;
	}

	if(cuLaunchGrid(func, 1 + (count-1)/16, 1 + (count-1)/16) != hipSuccess)
	{
		printf("ERROR: cuLaunchGrid (cudaBLAS2D)\n");
		return;
	}

	if(hipCtxSynchronize() != hipSuccess)
    {
        printf("ERROR: cudaContextSync (cudaBLAS2D)\n");
        return;
    }
}

JNIEXPORT void JNICALL Java_ppl_delite_cuda_DeliteCudaDriver_cudaBLAS1DAsync(JNIEnv *env, jobject obj, jlong func_id, jlong a_ptr, jlong b_ptr, jlong c_ptr, jint count, jlong stream)
{
	int offset = 0;
	void* ptr;
	hipFunction_t func = (hipFunction_t)func_id;

	ptr = (void*)(size_t)a_ptr;
	offset = (offset + __alignof(ptr) - 1) & ~(__alignof(ptr) - 1);

	if(cuParamSetv(func, offset, &ptr, sizeof(ptr)) != hipSuccess)
	{
		printf("ERROR: cuParamSetv (cudaBLAS1DAsync)\n");
		return;
	}
	offset += sizeof(ptr);

	ptr = (void*)(size_t)b_ptr;
	offset = (offset + __alignof(ptr) - 1) & ~(__alignof(ptr) - 1);
	if(cuParamSetv(func, offset, &ptr, sizeof(ptr)) != hipSuccess)
	{
		printf("ERROR: cuParamSetv (cudaBLAS1DAsync) \n");
		return;
	}
	offset += sizeof(ptr);

	ptr = (void*)(size_t)c_ptr;
	offset = (offset + __alignof(ptr) - 1) & ~(__alignof(ptr) - 1);
	if(cuParamSetv(func, offset, &ptr, sizeof(ptr)) != hipSuccess)
	{
		printf("ERROR: cuParamSetv (cudaBLAS1DAsync) \n");
		return;
	}
	offset += sizeof(ptr);

	offset = (offset + __alignof(count) - 1) & ~(__alignof(count) - 1);
	if(cuParamSeti(func, offset, count) != hipSuccess)
	{
		printf("ERROR: cuParamSetv (cudaBLAS1DAsync) \n");
		return;
	}
	offset += sizeof(count);

	if(cuParamSetSize(func, offset) != hipSuccess)
	{
		printf("ERROR: cuParamSetSize (cudaBLAS1DAsync)\n");
		return;
	}

	//TODO : Need to receive dimension info as inputs
	if(cuFuncSetBlockShape(func, 512, 1, 1) != hipSuccess)
	{
		printf("ERROR: cuFuncSetBlockShape (cudaBLAS1DAsync)\n");
		return;
	}

	if(cuLaunchGridAsync(func, 1+(count-1)/512, 1, (hipStream_t)stream) != hipSuccess)
	{
		printf("ERROR: cuLaunchGrid (cudaBLAS1DAsync)\n");
		return;
	}
	
}


JNIEXPORT void JNICALL Java_ppl_delite_cuda_DeliteCudaDriver_cudaBLASRandAsync(JNIEnv *env, jobject obj, jlong func_id, jlong a_ptr, jint count, jlong stream)
{
	int offset = 0;
	void* ptr;
	hipFunction_t func = (hipFunction_t)func_id;

	ptr = (void*)(size_t)a_ptr;
	offset = (offset + __alignof(ptr) - 1) & ~(__alignof(ptr) - 1);

	if(cuParamSetv(func, offset, &ptr, sizeof(ptr)) != hipSuccess)
	{
		printf("ERROR: cuParamSetv (cudaBLAS1DAsync)\n");
		return;
	}
	offset += sizeof(ptr);

	offset = (offset + __alignof(count) - 1) & ~(__alignof(count) - 1);
	if(cuParamSeti(func, offset, count) != hipSuccess)
	{
		printf("ERROR: cuParamSetv (cudaBLAS1DAsync) \n");
		return;
	}
	offset += sizeof(count);

	if(cuParamSetSize(func, offset) != hipSuccess)
	{
		printf("ERROR: cuParamSetSize (cudaBLAS1DAsync)\n");
		return;
	}

	//TODO : Need to receive dimension info as inputs
	if(cuFuncSetBlockShape(func, 512, 1, 1) != hipSuccess)
	{
		printf("ERROR: cuFuncSetBlockShape (cudaBLAS1DAsync)\n");
		return;
	}

	if(cuLaunchGridAsync(func, 1+(count-1)/512, 1, (hipStream_t)stream) != hipSuccess)
	{
		printf("ERROR: cuLaunchGrid (cudaBLAS1DAsync)\n");
		return;
	}

}

JNIEXPORT void JNICALL Java_ppl_delite_cuda_DeliteCudaDriver_cudaBLAS2DAsync(JNIEnv *env, jobject obj, jlong func_id, jlong a_ptr, jlong b_ptr, jlong c_ptr, jint count, jlong stream)
{
	int offset = 0;
	void* ptr;
	hipFunction_t func = (hipFunction_t)func_id;

	ptr = (void*)(size_t)a_ptr;
	offset = (offset + __alignof(ptr) - 1) & ~(__alignof(ptr) - 1);
	if(cuParamSetv(func, offset, &ptr, sizeof(ptr)) != hipSuccess)
	{
		printf("ERROR: cuParamSetv (cudaBLAS2DAsync) \n");
		return;
	}
	offset += sizeof(ptr);

	ptr = (void*)(size_t)b_ptr;
	offset = (offset + __alignof(ptr) - 1) & ~(__alignof(ptr) - 1); 
	if(cuParamSetv(func, offset, &ptr, sizeof(ptr)) != hipSuccess)
	{
		printf("ERROR: cuParamSetv (cudaBLAS2DAsync) \n");
		return;
	}
	offset += sizeof(ptr);

	ptr = (void*)(size_t)c_ptr;
	offset = (offset + __alignof(ptr) - 1) & ~(__alignof(ptr) - 1);
	if(cuParamSetv(func, offset, &ptr, sizeof(ptr)) != hipSuccess)
	{
		printf("ERROR: cuParamSetv (cudaBLAS2DAsync) \n");
		return;
	}
	offset += sizeof(ptr);

	offset = (offset + __alignof(count) - 1) & ~(__alignof(count) - 1);
	if(cuParamSeti(func, offset, count) != hipSuccess)
	{
		printf("ERROR: cuParamSetv (cudaBLAS2DAsync) \n");
		return;
	}
	offset += sizeof(count);

	if(cuParamSetSize(func, offset) != hipSuccess)
	{
		printf("ERROR: cuParamSetSize (cudaBLAS2DAsync) \n");
		return;
	}


	//TODO : Need to receive dimension info as inputs
	if(cuFuncSetBlockShape(func, 16, 1, 1) != hipSuccess)
	{
		printf("ERROR: cuFuncSetBlockShape (cudaBLAS2DAsync)\n");
		return;
	}

	if(cuLaunchGridAsync(func, 1+(count-1)/16, 1+(count-1)/16, (hipStream_t)stream) != hipSuccess)
	{
		printf("ERROR: cuLaunchGrid (cudaBLAS2DAsync)\n");
		return;
	}
	
}

JNIEXPORT void JNICALL Java_ppl_delite_cuda_DeliteCudaDriver_cudaMapAsync(JNIEnv *env, jobject obj, jlong func_id, jlong a_ptr, jlong b_ptr, jlong c_ptr, jint width, jint height, jint compareWith, jdouble weightedspamcount, jlong stream)
{
	int offset = 0;
	void* ptr;
	hipFunction_t func = (hipFunction_t)func_id;

	ptr = (void*)(size_t)a_ptr;
	offset = (offset + __alignof(ptr) - 1) & ~(__alignof(ptr) - 1);
	if(cuParamSetv(func, offset, &ptr, sizeof(ptr)) != hipSuccess)
	{
		printf("ERROR: cuParamSetv (cudaMapAsync)\n");
		return;
	}
	offset += sizeof(ptr);

	ptr = (void*)(size_t)b_ptr;
	offset = (offset + __alignof(ptr) - 1) & ~(__alignof(ptr) - 1); 
	if(cuParamSetv(func, offset, &ptr, sizeof(ptr)) != hipSuccess)
	{
		printf("ERROR: cuParamSetv (cudaMapAsync)\n");
		return;
	}
	offset += sizeof(ptr);

	ptr = (void*)(size_t)c_ptr;
	offset = (offset + __alignof(ptr) - 1) & ~(__alignof(ptr) - 1);
	if(cuParamSetv(func, offset, &ptr, sizeof(ptr)) != hipSuccess)
	{
		printf("ERROR: cuParamSetv (cudaMapAsync)\n");
		return;
	}
	offset += sizeof(ptr);

	offset = (offset + __alignof(width) - 1) & ~(__alignof(width) - 1);
	if(cuParamSeti(func, offset, width) != hipSuccess)
	{
		printf("ERROR: cuParamSetv (cudaMapAsync)\n");
		return;
	}
	offset += sizeof(width);
	
	offset = (offset + __alignof(height) - 1) & ~(__alignof(height) - 1);
	if(cuParamSeti(func, offset, height) != hipSuccess)
	{
		printf("ERROR: cuParamSetv (cudaMapAsync)\n");
		return;
	}
	offset += sizeof(height);
	
	offset = (offset + __alignof(compareWith) - 1) & ~(__alignof(compareWith) - 1);
	if(cuParamSeti(func, offset, compareWith) != hipSuccess)
	{
		printf("ERROR: cuParamSetv (cudaMapAsync)\n");
		return;
	}
	offset += sizeof(compareWith);

	offset = (offset + __alignof(weightedspamcount) - 1) & ~(__alignof(weightedspamcount) - 1);
	if(cuParamSetv(func, offset, &weightedspamcount, sizeof(weightedspamcount)) != hipSuccess)
	{
		printf("ERROR: cuParamSetv (cudaMapAsync)\n");
		return;
	}
	offset += sizeof(weightedspamcount);


	if(cuParamSetSize(func, offset) != hipSuccess)
	{
		printf("ERROR: cuParamSetSize (cudaMapAsync)\n");
		return;
	}


	//TODO : Need to receive dimension info as inputs
	if(cuFuncSetBlockShape(func, 16, 16, 1) != hipSuccess)
	{
		printf("ERROR: cuFuncSetBlockShape (cudaMapAsync)\n");
		return;
	}

	if(cuLaunchGridAsync(func, 1+(width-1)/16, 1, (hipStream_t)stream) != hipSuccess)
	{
		printf("ERROR: cuLaunchGrid (cudaMapAsync)\n");
		return;
	}
}

JNIEXPORT void JNICALL Java_ppl_delite_cuda_DeliteCudaDriver_cudaMapToVecAsync(JNIEnv *env, jobject obj, jlong func_id, jlong a_ptr, jlong b_ptr, jint width, jint height, jlong stream)
{
	int offset = 0;
	void* ptr;
	hipFunction_t func = (hipFunction_t)func_id;

	ptr = (void*)(size_t)a_ptr;
	offset = (offset + __alignof(ptr) - 1) & ~(__alignof(ptr) - 1);
	if(cuParamSetv(func, offset, &ptr, sizeof(ptr)) != hipSuccess)
	{
		printf("ERROR: cuParamSetv (cudaMapToVecAsync)\n");
		return;
	}
	offset += sizeof(ptr);

	ptr = (void*)(size_t)b_ptr;
	offset = (offset + __alignof(ptr) - 1) & ~(__alignof(ptr) - 1); 
	if(cuParamSetv(func, offset, &ptr, sizeof(ptr)) != hipSuccess)
	{
		printf("ERROR: cuParamSetv (cudaMapToVecAsync)\n");
		return;
	}
	offset += sizeof(ptr);

	offset = (offset + __alignof(width) - 1) & ~(__alignof(width) - 1);
	if(cuParamSeti(func, offset, width) != hipSuccess)
	{
		printf("ERROR: cuParamSetv (cudaMapToVecAsync)\n");
		return;
	}
	offset += sizeof(width);
	
	offset = (offset + __alignof(height) - 1) & ~(__alignof(height) - 1);
	if(cuParamSeti(func, offset, height) != hipSuccess)
	{
		printf("ERROR: cuParamSetv (cudaMapToVecAsync)\n");
		return;
	}
	offset += sizeof(height);
	
	if(cuParamSetSize(func, offset) != hipSuccess)
	{
		printf("ERROR: cuParamSetSize (cudaMapToVecAsync)\n");
		return;
	}


	//TODO : Need to receive dimension info as inputs
	if(cuFuncSetBlockShape(func, 256, 1, 1) != hipSuccess)
	{
		printf("ERROR: cuFuncSetBlockShape (cudaMapToVecAsync)\n");
		return;
	}

	if(cuLaunchGridAsync(func, 1+(width-1)/256, 1, (hipStream_t)stream) != hipSuccess)
	{
		printf("ERROR: cuLaunchGrid (cudaMapToVecAsync)\n");
		return;
	}
	
}

JNIEXPORT void JNICALL Java_ppl_delite_cuda_DeliteCudaDriver_cudaMapLRAsync(JNIEnv *env, jobject obj, jlong func_id, jlong a_ptr, jlong b_ptr, jdouble x_cur, jint tau, jint count, jlong stream)
{
	int offset = 0;
	void* ptr;
	hipFunction_t func = (hipFunction_t)func_id;

	ptr = (void*)(size_t)a_ptr;
	offset = (offset + __alignof(ptr) - 1) & ~(__alignof(ptr) - 1);
	if(cuParamSetv(func, offset, &ptr, sizeof(ptr)) != hipSuccess)
	{
		printf("ERROR: cuParamSetv (cudaMapLRAsync)\n");
		return;
	}
	offset += sizeof(ptr);

	ptr = (void*)(size_t)b_ptr;
	offset = (offset + __alignof(ptr) - 1) & ~(__alignof(ptr) - 1); 
	if(cuParamSetv(func, offset, &ptr, sizeof(ptr)) != hipSuccess)
	{
		printf("ERROR: cuParamSetv (cudaMapLRAsync)\n");
		return;
	}
	offset += sizeof(ptr);

	offset = (offset + __alignof(x_cur) - 1) & ~(__alignof(x_cur) - 1);
	if(cuParamSetv(func, offset, &x_cur, sizeof(x_cur)) != hipSuccess)
	{
		printf("ERROR: cuParamSetv (cudaMapLRAsync)\n");
		return;
	}
	offset += sizeof(x_cur);
	
	offset = (offset + __alignof(tau) - 1) & ~(__alignof(tau) - 1);
	if(cuParamSeti(func, offset, tau) != hipSuccess)
	{
		printf("ERROR: cuParamSetv (cudaMapLRAsync)\n");
		return;
	}
	offset += sizeof(tau);
	
	offset = (offset + __alignof(count) - 1) & ~(__alignof(count) - 1);
	if(cuParamSeti(func, offset, count) != hipSuccess)
	{
		printf("ERROR: cuParamSetv (cudaMapLRAsync)\n");
		return;
	}
	offset += sizeof(count);
	
	if(cuParamSetSize(func, offset) != hipSuccess)
	{
		printf("ERROR: cuParamSetSize (cudaMapLRAsync)\n");
		return;
	}


	//TODO : Need to receive dimension info as inputs
	if(cuFuncSetBlockShape(func, 32, 1, 1) != hipSuccess)
	{
		printf("ERROR: cuFuncSetBlockShape (cudaMapLRAsync)\n");
		return;
	}

	if(cuLaunchGridAsync(func, 1+(count-1)/32, 1, (hipStream_t)stream) != hipSuccess)
	{
		printf("ERROR: cuLaunchGrid (cudaMapLRAsync)\n");
		return;
	}
}


JNIEXPORT void JNICALL Java_ppl_delite_cuda_DeliteCudaDriver_cudaAsync3D2I(JNIEnv *env, jobject obj, jlong func_id, jlong a_ptr, jlong b_ptr, jlong c_ptr, jint width, jint height, jint numTB, jlong stream)
{
	int offset = 0;
	void* ptr;
	hipFunction_t func = (hipFunction_t)func_id;

	ptr = (void*)(size_t)a_ptr;
	offset = (offset + __alignof(ptr) - 1) & ~(__alignof(ptr) - 1);
	if(cuParamSetv(func, offset, &ptr, sizeof(ptr)) != hipSuccess)
	{
		printf("ERROR: cuParamSetv (cudaAsync3D2I)\n");
		return;
	}
	offset += sizeof(ptr);

	ptr = (void*)(size_t)b_ptr;
	offset = (offset + __alignof(ptr) - 1) & ~(__alignof(ptr) - 1); 
	if(cuParamSetv(func, offset, &ptr, sizeof(ptr)) != hipSuccess)
	{
		printf("ERROR: cuParamSetv (cudaAsync3D2I)\n");
		return;
	}
	offset += sizeof(ptr);
	
	ptr = (void*)(size_t)c_ptr;
	offset = (offset + __alignof(ptr) - 1) & ~(__alignof(ptr) - 1); 
	if(cuParamSetv(func, offset, &ptr, sizeof(ptr)) != hipSuccess)
	{
		printf("ERROR: cuParamSetv (cudaAsync3D2I)\n");
		return;
	}
	offset += sizeof(ptr);

	offset = (offset + __alignof(width) - 1) & ~(__alignof(width) - 1);
	if(cuParamSeti(func, offset, width) != hipSuccess)
	{
		printf("ERROR: cuParamSetv (cudaAsync3D2I)\n");
		return;
	}
	offset += sizeof(width);
	
	offset = (offset + __alignof(height) - 1) & ~(__alignof(height) - 1);
	if(cuParamSeti(func, offset, height) != hipSuccess)
	{
		printf("ERROR: cuParamSetv (cudaAsync3D2I)\n");
		return;
	}
	offset += sizeof(height);
	
	if(cuParamSetSize(func, offset) != hipSuccess)
	{
		printf("ERROR: cuParamSetSize (cudaAsync3D2I)\n");
		return;
	}


	//TODO : Need to receive dimension info as inputs
	if(cuFuncSetBlockShape(func, 256, 1, 1) != hipSuccess)
	{
		printf("ERROR: cuFuncSetBlockShape (cudaAsync3D2I)\n");
		return;
	}

	if(cuLaunchGridAsync(func, 1+(numTB-1)/256, 1, (hipStream_t)stream) != hipSuccess)
	{
		printf("ERROR: cuLaunchGrid (cudaAsync3D2I)\n");
		return;
	}
}


JNIEXPORT void JNICALL Java_ppl_delite_cuda_DeliteCudaDriver_cudaAsync2D1D1I(JNIEnv *env, jobject obj, jlong func_id, jlong a_ptr, jlong b_ptr, jdouble dvalue, jint length, jlong stream)
{
	int offset = 0;
	void* ptr;
	hipFunction_t func = (hipFunction_t)func_id;

	ptr = (void*)(size_t)a_ptr;
	offset = (offset + __alignof(ptr) - 1) & ~(__alignof(ptr) - 1);
	if(cuParamSetv(func, offset, &ptr, sizeof(ptr)) != hipSuccess)
	{
		printf("ERROR: cuParamSetv (cudaAsync2D1D1I) \n");
		return;
	}
	offset += sizeof(ptr);

	ptr = (void*)(size_t)b_ptr;
	offset = (offset + __alignof(ptr) - 1) & ~(__alignof(ptr) - 1); 
	if(cuParamSetv(func, offset, &ptr, sizeof(ptr)) != hipSuccess)
	{
		printf("ERROR: cuParamSetv (cudaAsync2D1D1I) \n");
		return;
	}
	offset += sizeof(ptr);
	
	offset = (offset + __alignof(dvalue) - 1) & ~(__alignof(dvalue) - 1);
	if(cuParamSetv(func, offset, &dvalue, sizeof(dvalue)) != hipSuccess)
	{
		printf("ERROR: cuParamSetv (cudaAsync2D1D1I)\n");
		return;
	}
	offset += sizeof(dvalue);
	
	offset = (offset + __alignof(length) - 1) & ~(__alignof(length) - 1);
	if(cuParamSeti(func, offset, length) != hipSuccess)
	{
		printf("ERROR: cuParamSetv (cudaAsync2D1D1I)\n");
		return;
	}
	offset += sizeof(length);
	
	if(cuParamSetSize(func, offset) != hipSuccess)
	{
		printf("ERROR: cuParamSetSize (cudaAsync2D1D1I)\n");
		return;
	}


	//TODO : Need to receive dimension info as inputs
	if(cuFuncSetBlockShape(func, 256, 1, 1) != hipSuccess)
	{
		printf("ERROR: cuFuncSetBlockShape (cudaAsync2D1D1I)\n");
		return;
	}

	if(cuLaunchGridAsync(func, 1+(length-1)/256, 1, (hipStream_t)stream) != hipSuccess)
	{
		printf("ERROR: cuLaunchGrid (cudaAsync2D1D1I)\n");
		return;
	}
	
}

JNIEXPORT void JNICALL Java_ppl_delite_cuda_DeliteCudaDriver_cudaAsync2D2I2Dim(JNIEnv *env, jobject obj, jlong func_id, jlong a_ptr, jlong b_ptr, jint width, jint height, jlong stream)
{
	int offset = 0;
	void* ptr;
	hipFunction_t func = (hipFunction_t)func_id;

	ptr = (void*)(size_t)a_ptr;
	offset = (offset + __alignof(ptr) - 1) & ~(__alignof(ptr) - 1);
	if(cuParamSetv(func, offset, &ptr, sizeof(ptr)) != hipSuccess)
	{
		printf("ERROR: cuParamSetv (cudaAsync2D2I2Dim)\n");
		return;
	}
	offset += sizeof(ptr);

	ptr = (void*)(size_t)b_ptr;
	offset = (offset + __alignof(ptr) - 1) & ~(__alignof(ptr) - 1); 
	if(cuParamSetv(func, offset, &ptr, sizeof(ptr)) != hipSuccess)
	{
		printf("ERROR: cuParamSetv (cudaAsync2D2I2Dim)\n");
		return;
	}
	offset += sizeof(ptr);
	
	offset = (offset + __alignof(width) - 1) & ~(__alignof(width) - 1);
	if(cuParamSeti(func, offset, width) != hipSuccess)
	{
		printf("ERROR: cuParamSetv (cudaAsync2D2I2Dim)\n");
		return;
	}
	offset += sizeof(width);
	
	offset = (offset + __alignof(height) - 1) & ~(__alignof(height) - 1);
	if(cuParamSeti(func, offset, height) != hipSuccess)
	{
		printf("ERROR: cuParamSetv (cudaAsync2D2I2Dim)\n");
		return;
	}
	offset += sizeof(height);
	
	if(cuParamSetSize(func, offset) != hipSuccess)
	{
		printf("ERROR: cuParamSetSize (cudaAsync2D2I2Dim)\n");
		return;
	}


	//TODO : Need to receive dimension info as inputs
	if(cuFuncSetBlockShape(func, 16, 16, 1) != hipSuccess)
	{
		printf("ERROR: cuFuncSetBlockShape (cudaAsync2D2I2Dim)\n");
		return;
	}

	if(cuLaunchGridAsync(func, 1+(width-1)/16, 1+(height-1)/16, (hipStream_t)stream) != hipSuccess)
	{
		printf("ERROR: cuLaunchGrid (cudaAsync2D2I2Dim)\n");
		return;
	}
	
}

*/
/*******************************
** RBM calls
*********************************/
/*
JNIEXPORT void JNICALL Java_ppl_delite_cuda_DeliteCudaDriver_cudaAsyncRBM_11I(JNIEnv *env, jobject obj, jlong func_id, jlong a_ptr, jlong c_ptr, jint length, jlong stream)
{
	int offset = 0;
	void* ptr;
	hipFunction_t func = (hipFunction_t)func_id;

	ptr = (void*)(size_t)a_ptr;
	offset = (offset + __alignof(ptr) - 1) & ~(__alignof(ptr) - 1);
	if(cuParamSetv(func, offset, &ptr, sizeof(ptr)) != hipSuccess)
	{
		printf("ERROR: cuParamSetv (cudaAsyncRBM_11I)\n");
		return;
	}
	offset += sizeof(ptr);

	ptr = (void*)(size_t)c_ptr;
	offset = (offset + __alignof(ptr) - 1) & ~(__alignof(ptr) - 1);
	if(cuParamSetv(func, offset, &ptr, sizeof(ptr)) != hipSuccess)
	{
		printf("ERROR: cuParamSetv (cudaAsyncRBM_11I)\n");
		return;
	}
	offset += sizeof(ptr);

	offset = (offset + __alignof(length) - 1) & ~(__alignof(length) - 1);
	if(cuParamSeti(func, offset, length) != hipSuccess)
	{
		printf("ERROR: cuParamSetv (cudaAsyncRBM_11I)\n");
		return;
	}
	offset += sizeof(length);

	if(cuParamSetSize(func, offset) != hipSuccess)
	{
		printf("ERROR: cuParamSetSize (cudaAsyncRBM_11I)\n");
		return;
	}
	
	if(cuFuncSetBlockShape(func, 512, 1, 1) != hipSuccess)
	{
		printf("ERROR: cuFuncSetBlockShape (cudaAsyncRBM_11I)\n");
		return;
	}

	if(cuLaunchGridAsync(func, 1+(length-1)/512, 1, (hipStream_t)stream) != hipSuccess)
	{
		printf("ERROR: cuLaunchGrid (cudaAsyncRBM_11I)\n");
		return;
	}
}

JNIEXPORT void JNICALL Java_ppl_delite_cuda_DeliteCudaDriver_cudaAsyncRBM_11I1S(JNIEnv *env, jobject obj, jlong func_id, jlong a_ptr, jlong c_ptr, jdouble svalue, jint length, jlong stream)
{
	int offset = 0;
	void* ptr;
	hipFunction_t func = (hipFunction_t)func_id;

	ptr = (void*)(size_t)a_ptr;
	offset = (offset + __alignof(ptr) - 1) & ~(__alignof(ptr) - 1);
	if(cuParamSetv(func, offset, &ptr, sizeof(ptr)) != hipSuccess)
	{
		printf("ERROR: cuParamSetv (cudaAsyncRBM_11I1S)\n");
		return;
	}
	offset += sizeof(ptr);

	ptr = (void*)(size_t)c_ptr;
	offset = (offset + __alignof(ptr) - 1) & ~(__alignof(ptr) - 1);
	if(cuParamSetv(func, offset, &ptr, sizeof(ptr)) != hipSuccess)
	{
		printf("ERROR: cuParamSetv (cudaAsyncRBM_11I1S)\n");
		return;
	}
	offset += sizeof(ptr);

    offset = (offset + __alignof(svalue) - 1) & ~(__alignof(svalue) - 1);
	if(cuParamSetv(func, offset, &svalue, sizeof(svalue)) != hipSuccess)
	{
		printf("ERROR: cuParamSetv (cudaAsyncRBM_11I1S)\n");
		return;
	}
	offset += sizeof(svalue);

	offset = (offset + __alignof(length) - 1) & ~(__alignof(length) - 1);
	if(cuParamSeti(func, offset, length) != hipSuccess)
	{
		printf("ERROR: cuParamSetv (cudaAsyncRBM_11I1S)\n");
		return;
	}
	offset += sizeof(length);

	if(cuParamSetSize(func, offset) != hipSuccess)
	{
		printf("ERROR: cuParamSetSize (cudaAsyncRBM_11I1S)\n");
		return;
	}

	if(cuFuncSetBlockShape(func, 512, 1, 1) != hipSuccess)
	{
		printf("ERROR: cuFuncSetBlockShape (cudaAsyncRBM_11I1S)\n");
		return;
	}

	if(cuLaunchGridAsync(func, 1+(length-1)/512, 1, (hipStream_t)stream) != hipSuccess)
	{
		printf("ERROR: cuLaunchGrid (cudaAsyncRBM_11I1S)\n");
		return;
	}
}
                  
JNIEXPORT void JNICALL Java_ppl_delite_cuda_DeliteCudaDriver_cudaAsyncRBM_11I1S_1Float(JNIEnv *env, jobject obj, jlong func_id, jlong a_ptr, jlong c_ptr, jfloat svalue, jint length, jlong stream)
{
	int offset = 0;
	void* ptr;
	hipFunction_t func = (hipFunction_t)func_id;

	ptr = (void*)(size_t)a_ptr;
	offset = (offset + __alignof(ptr) - 1) & ~(__alignof(ptr) - 1);
	if(cuParamSetv(func, offset, &ptr, sizeof(ptr)) != hipSuccess)
	{
		printf("ERROR: cuParamSetv (cudaAsyncRBM_11I1S)\n");
		return;
	}
	offset += sizeof(ptr);

	ptr = (void*)(size_t)c_ptr;
	offset = (offset + __alignof(ptr) - 1) & ~(__alignof(ptr) - 1);
	if(cuParamSetv(func, offset, &ptr, sizeof(ptr)) != hipSuccess)
	{
		printf("ERROR: cuParamSetv (cudaAsyncRBM_11I1S)\n");
		return;
	}
	offset += sizeof(ptr);

    offset = (offset + __alignof(svalue) - 1) & ~(__alignof(svalue) - 1);
	if(cuParamSetv(func, offset, &svalue, sizeof(svalue)) != hipSuccess)
	{
		printf("ERROR: cuParamSetv (cudaAsyncRBM_11I1S)\n");
		return;
	}
	offset += sizeof(svalue);

	offset = (offset + __alignof(length) - 1) & ~(__alignof(length) - 1);
	if(cuParamSeti(func, offset, length) != hipSuccess)
	{
		printf("ERROR: cuParamSetv (cudaAsyncRBM_11I1S)\n");
		return;
	}
	offset += sizeof(length);

	if(cuParamSetSize(func, offset) != hipSuccess)
	{
		printf("ERROR: cuParamSetSize (cudaAsyncRBM_11I1S)\n");
		return;
	}

	if(cuFuncSetBlockShape(func, 512, 1, 1) != hipSuccess)
	{
		printf("ERROR: cuFuncSetBlockShape (cudaAsyncRBM_11I1S)\n");
		return;
	}

	if(cuLaunchGridAsync(func, 1+(length-1)/512, 1, (hipStream_t)stream) != hipSuccess)
	{
		printf("ERROR: cuLaunchGrid (cudaAsyncRBM_11I1S)\n");
		return;
	}
}

JNIEXPORT void JNICALL Java_ppl_delite_cuda_DeliteCudaDriver_cudaAsyncRBM_12I(JNIEnv *env, jobject obj, jlong func_id, jlong a_ptr, jlong b_ptr, jlong c_ptr, jint length, jlong stream)
{
	int offset = 0;
	void* ptr;
	hipFunction_t func = (hipFunction_t)func_id;

	ptr = (void*)(size_t)a_ptr;
	offset = (offset + __alignof(ptr) - 1) & ~(__alignof(ptr) - 1);
	if(cuParamSetv(func, offset, &ptr, sizeof(ptr)) != hipSuccess)
	{
		printf("ERROR: cuParamSetv (cudaAsyncRBM_12I)\n");
		return;
	}
	offset += sizeof(ptr);

	ptr = (void*)(size_t)b_ptr;
	offset = (offset + __alignof(ptr) - 1) & ~(__alignof(ptr) - 1);
	if(cuParamSetv(func, offset, &ptr, sizeof(ptr)) != hipSuccess)
	{
		printf("ERROR: cuParamSetv (cudaAsyncRBM_12I)\n");
		return;
	}
	offset += sizeof(ptr);

    ptr = (void*)(size_t)c_ptr;
	offset = (offset + __alignof(ptr) - 1) & ~(__alignof(ptr) - 1);
	if(cuParamSetv(func, offset, &ptr, sizeof(ptr)) != hipSuccess)
	{
		printf("ERROR: cuParamSetv (cudaAsyncRBM_12I)\n");
		return;
	}
	offset += sizeof(ptr);

	offset = (offset + __alignof(length) - 1) & ~(__alignof(length) - 1);
	if(cuParamSeti(func, offset, length) != hipSuccess)
	{
		printf("ERROR: cuParamSetv (cudaAsyncRBM_12I)\n");
		return;
	}
	offset += sizeof(length);

	if(cuParamSetSize(func, offset) != hipSuccess)
	{
		printf("ERROR: cuParamSetSize (cudaAsyncRBM_12I)\n");
		return;
	}

	if(cuFuncSetBlockShape(func, 512, 1, 1) != hipSuccess)
	{
		printf("ERROR: cuFuncSetBlockShape (cudaAsyncRBM_12I)\n");
		return;
	}

	if(cuLaunchGridAsync(func, 1+(length-1)/512, 1, (hipStream_t)stream) != hipSuccess)
	{
		printf("ERROR: cuLaunchGrid (cudaAsyncRBM_12I)\n");
		return;
	}
}


JNIEXPORT void JNICALL Java_ppl_delite_cuda_DeliteCudaDriver_cudaAsyncRBM_1Repmat(JNIEnv *env, jobject obj, jlong func_id, jlong a_ptr, jlong c_ptr, jint length, int repRow, int repCol, jlong stream)
{
	int offset = 0;
	void* ptr;
	hipFunction_t func = (hipFunction_t)func_id;

	ptr = (void*)(size_t)a_ptr;
	offset = (offset + __alignof(ptr) - 1) & ~(__alignof(ptr) - 1);
	if(cuParamSetv(func, offset, &ptr, sizeof(ptr)) != hipSuccess)
	{
		printf("ERROR: cuParamSetv (cudaAsyncRBM_1Repmat)\n");
		return;
	}
	offset += sizeof(ptr);

    ptr = (void*)(size_t)c_ptr;
	offset = (offset + __alignof(ptr) - 1) & ~(__alignof(ptr) - 1);
	if(cuParamSetv(func, offset, &ptr, sizeof(ptr)) != hipSuccess)
	{
		printf("ERROR: cuParamSetv (cudaAsyncRBM_1Repmat)\n");
		return;
	}
	offset += sizeof(ptr);

	offset = (offset + __alignof(length) - 1) & ~(__alignof(length) - 1);
	if(cuParamSeti(func, offset, length) != hipSuccess)
	{
		printf("ERROR: cuParamSetv (cudaAsyncRBM_1Repmat)\n");
		return;
	}
	offset += sizeof(length);

    offset = (offset + __alignof(repRow) - 1) & ~(__alignof(repRow) - 1);
	if(cuParamSeti(func, offset, repRow) != hipSuccess)
	{
		printf("ERROR: cuParamSetv (cudaAsyncRBM_1Repmat)\n");
		return;
	}
	offset += sizeof(repRow);

	offset = (offset + __alignof(repCol) - 1) & ~(__alignof(repCol) - 1);
	if(cuParamSeti(func, offset, repCol) != hipSuccess)
	{
		printf("ERROR: cuParamSetv (cudaAsyncRBM_1Repmat)\n");
		return;
	}
	offset += sizeof(repCol);

	if(cuParamSetSize(func, offset) != hipSuccess)
	{
		printf("ERROR: cuParamSetSize (cudaAsyncRBM_1Repmat)\n");
		return;
	}

	if(cuFuncSetBlockShape(func, 512, 1, 1) != hipSuccess)
	{
		printf("ERROR: cuFuncSetBlockShape (cudaAsyncRBM_1Repmat)\n");
		return;
	}

	if(cuLaunchGridAsync(func, 1+(length*repCol-1)/512, repRow, (hipStream_t)stream) != hipSuccess)
	{
		printf("ERROR: cuLaunchGrid (cudaAsyncRBM_1Repmat)\n");
		return;
	}
}


JNIEXPORT void JNICALL Java_ppl_delite_cuda_DeliteCudaDriver_cudaAsyncRBM_11I2D(JNIEnv *env, jobject obj, jlong func_id, jlong a_ptr, jlong c_ptr, jint numRows, jint numCols, jlong stream)
{
	int offset = 0;
	void* ptr;
	hipFunction_t func = (hipFunction_t)func_id;

	ptr = (void*)(size_t)a_ptr;
	offset = (offset + __alignof(ptr) - 1) & ~(__alignof(ptr) - 1);
	if(cuParamSetv(func, offset, &ptr, sizeof(ptr)) != hipSuccess)
	{
		printf("ERROR: cuParamSetv (cudaAsyncRBM_11I2D)\n");
		return;
	}
	offset += sizeof(ptr);

	ptr = (void*)(size_t)c_ptr;
	offset = (offset + __alignof(ptr) - 1) & ~(__alignof(ptr) - 1);
	if(cuParamSetv(func, offset, &ptr, sizeof(ptr)) != hipSuccess)
	{
		printf("ERROR: cuParamSetv (cudaAsyncRBM_11I2D)\n");
		return;
	}
	offset += sizeof(ptr);

	offset = (offset + __alignof(numRows) - 1) & ~(__alignof(numRows) - 1);
	if(cuParamSeti(func, offset, numRows) != hipSuccess)
	{
		printf("ERROR: cuParamSetv (cudaAsyncRBM_11I2D)\n");
		return;
	}
	offset += sizeof(numRows);

	offset = (offset + __alignof(numCols) - 1) & ~(__alignof(numCols) - 1);
	if(cuParamSeti(func, offset, numCols) != hipSuccess)
	{
		printf("ERROR: cuParamSetv (cudaAsyncRBM_11I2D)\n");
		return;
	}
	offset += sizeof(numCols);

	if(cuParamSetSize(func, offset) != hipSuccess)
	{
		printf("ERROR: cuParamSetSize (cudaAsyncRBM_11I2D)\n");
		return;
	}

	if(cuFuncSetBlockShape(func, 512, 1, 1) != hipSuccess)
	{
		printf("ERROR: cuFuncSetBlockShape (cudaAsyncRBM_11I2D)\n");
		return;
	}

	if(cuLaunchGridAsync(func, 1+(numCols-1)/512, 1, (hipStream_t)stream) != hipSuccess)
	{
		printf("ERROR: cuLaunchGrid (cudaAsyncRBM_11I2D)\n");
		return;
	}
}


JNIEXPORT void JNICALL Java_ppl_delite_cuda_DeliteCudaDriver_cudaAsyncRBM_11I3D(JNIEnv *env, jobject obj, jlong func_id, jlong a_ptr, jlong b_ptr, jlong c_ptr, jint numRows, jint numCols, jlong stream)
{
	int offset = 0;
	void* ptr;
	hipFunction_t func = (hipFunction_t)func_id;

	ptr = (void*)(size_t)a_ptr;
	offset = (offset + __alignof(ptr) - 1) & ~(__alignof(ptr) - 1);
	if(cuParamSetv(func, offset, &ptr, sizeof(ptr)) != hipSuccess)
	{
		printf("ERROR: cuParamSetv (cudaAsyncRBM_11I3D)\n");
		return;
	}
	offset += sizeof(ptr);

    ptr = (void*)(size_t)b_ptr;
	offset = (offset + __alignof(ptr) - 1) & ~(__alignof(ptr) - 1);
	if(cuParamSetv(func, offset, &ptr, sizeof(ptr)) != hipSuccess)
	{
		printf("ERROR: cuParamSetv (cudaAsyncRBM_11I3D)\n");
		return;
	}
	offset += sizeof(ptr);

	ptr = (void*)(size_t)c_ptr;
	offset = (offset + __alignof(ptr) - 1) & ~(__alignof(ptr) - 1);
	if(cuParamSetv(func, offset, &ptr, sizeof(ptr)) != hipSuccess)
	{
		printf("ERROR: cuParamSetv (cudaAsyncRBM_11I3D)\n");
		return;
	}
	offset += sizeof(ptr);

	offset = (offset + __alignof(numRows) - 1) & ~(__alignof(numRows) - 1);
	if(cuParamSeti(func, offset, numRows) != hipSuccess)
	{
		printf("ERROR: cuParamSetv (cudaAsyncRBM_11I3D)\n");
		return;
	}
	offset += sizeof(numRows);

	offset = (offset + __alignof(numCols) - 1) & ~(__alignof(numCols) - 1);
	if(cuParamSeti(func, offset, numCols) != hipSuccess)
	{
		printf("ERROR: cuParamSetv (cudaAsyncRBM_11I3D)\n");
		return;
	}
	offset += sizeof(numCols);

	if(cuParamSetSize(func, offset) != hipSuccess)
	{
		printf("ERROR: cuParamSetSize (cudaAsyncRBM_11I3D)\n");
		return;
	}

	if(cuFuncSetBlockShape(func, 512, 1, 1) != hipSuccess)
	{
		printf("ERROR: cuFuncSetBlockShape (cudaAsyncRBM_11I3D)\n");
		return;
	}

	if(cuLaunchGridAsync(func, 1+(numCols-1)/512, 1, (hipStream_t)stream) != hipSuccess)
	{
		printf("ERROR: cuLaunchGrid (cudaAsyncRBM_11I3D)\n");
		return;
	}
}
*/

/*
// K-means
JNIEXPORT void JNICALL Java_ppl_delite_cuda_DeliteCudaDriver_cudaAsyncKM1(JNIEnv *env, jobject obj, jlong func_id, jlong x_ptr, jlong mu_ptr, jlong out_ptr, jint x_numRows, jint x_numCols, jint mu_numRows, jint mu_numCols, jlong stream)
{
	int offset = 0;
	void* ptr;
	hipFunction_t func = (hipFunction_t)func_id;

	ptr = (void*)(size_t)x_ptr;
	offset = (offset + __alignof(ptr) - 1) & ~(__alignof(ptr) - 1);
	if(cuParamSetv(func, offset, &ptr, sizeof(ptr)) != hipSuccess)
	{
		printf("ERROR: cuParamSetv (cudaAsyncKM)\n");
		return;
	}
	offset += sizeof(ptr);

	ptr = (void*)(size_t)mu_ptr;
	offset = (offset + __alignof(ptr) - 1) & ~(__alignof(ptr) - 1);
	if(cuParamSetv(func, offset, &ptr, sizeof(ptr)) != hipSuccess)
	{
		printf("ERROR: cuParamSetv (cudaAsyncKM)\n");
		return;
	}
	offset += sizeof(ptr);

    ptr = (void*)(size_t)out_ptr;
	offset = (offset + __alignof(ptr) - 1) & ~(__alignof(ptr) - 1);
	if(cuParamSetv(func, offset, &ptr, sizeof(ptr)) != hipSuccess)
	{
		printf("ERROR: cuParamSetv (cudaAsyncKM)\n");
		return;
	}
	offset += sizeof(ptr);

	offset = (offset + __alignof(x_numRows) - 1) & ~(__alignof(x_numRows) - 1);
	if(cuParamSeti(func, offset, x_numRows) != hipSuccess)
	{
		printf("ERROR: cuParamSetv (cudaAsyncKM)\n");
		return;
	}
	offset += sizeof(x_numRows);

	offset = (offset + __alignof(x_numCols) - 1) & ~(__alignof(x_numCols) - 1);
	if(cuParamSeti(func, offset, x_numCols) != hipSuccess)
	{
		printf("ERROR: cuParamSetv (cudaAsyncKM)\n");
		return;
	}
	offset += sizeof(x_numCols);

	offset = (offset + __alignof(mu_numRows) - 1) & ~(__alignof(mu_numRows) - 1);
	if(cuParamSeti(func, offset, mu_numRows) != hipSuccess)
	{
		printf("ERROR: cuParamSetv (cudaAsyncKM)\n");
		return;
	}
	offset += sizeof(mu_numRows);

	offset = (offset + __alignof(mu_numCols) - 1) & ~(__alignof(mu_numCols) - 1);
	if(cuParamSeti(func, offset, mu_numCols) != hipSuccess)
	{
		printf("ERROR: cuParamSetv (cudaAsyncKM)\n");
		return;
	}
	offset += sizeof(mu_numCols);

	if(cuParamSetSize(func, offset) != hipSuccess)
	{
		printf("ERROR: cuParamSetSize (cudaAsyncKM)\n");
		return;
	}


	//if(cuFuncSetBlockShape(func, x_numCols, 1, 1) != hipSuccess)
	//{
	//	printf("ERROR: cuFuncSetBlockShape (cudaAsyncKM)\n");
	//	return;
	//}
	//if(cuLaunchGridAsync(func, 1, x_numRows, (hipStream_t)stream) != hipSuccess)
	//{
	//	printf("ERROR: cuLaunchGrid (cudaAsyncKM)\n");
	//	return;
	//}
	int dimy = 512/x_numCols;

	if(cuFuncSetBlockShape(func, x_numCols, dimy, 1) != hipSuccess)
	{
		printf("ERROR: cuFuncSetBlockShape (cudaAsyncKM)\n");
		return;
	}
	if(cuLaunchGridAsync(func, 1, 1+(x_numRows-1)/dimy, (hipStream_t)stream) != hipSuccess)
	{
		printf("ERROR: cuLaunchGrid (cudaAsyncKM)\n");
		return;
	}
}

*/
/*
JNIEXPORT void JNICALL Java_ppl_delite_cuda_DeliteCudaDriver_cudaAsyncKM2(JNIEnv *env, jobject obj, jlong func_id, jlong x_ptr, jlong mu_ptr, jlong c_ptr, jint x_numRows, jint x_numCols, jint mu_numRows, jlong stream)
{
	int offset = 0;
	void* ptr;
	hipFunction_t func = (hipFunction_t)func_id;

	ptr = (void*)(size_t)x_ptr;
	offset = (offset + __alignof(ptr) - 1) & ~(__alignof(ptr) - 1);
	if(cuParamSetv(func, offset, &ptr, sizeof(ptr)) != hipSuccess)
	{
		printf("ERROR: cuParamSetv (cudaAsyncKM2)\n");
		return;
	}
	offset += sizeof(ptr);

	ptr = (void*)(size_t)mu_ptr;
	offset = (offset + __alignof(ptr) - 1) & ~(__alignof(ptr) - 1);
	if(cuParamSetv(func, offset, &ptr, sizeof(ptr)) != hipSuccess)
	{
		printf("ERROR: cuParamSetv (cudaAsyncKM2)\n");
		return;
	}
	offset += sizeof(ptr);

    ptr = (void*)(size_t)c_ptr;
	offset = (offset + __alignof(ptr) - 1) & ~(__alignof(ptr) - 1);
	if(cuParamSetv(func, offset, &ptr, sizeof(ptr)) != hipSuccess)
	{
		printf("ERROR: cuParamSetv (cudaAsyncKM2)\n");
		return;
	}
	offset += sizeof(ptr);

	offset = (offset + __alignof(x_numRows) - 1) & ~(__alignof(x_numRows) - 1);
	if(cuParamSeti(func, offset, x_numRows) != hipSuccess)
	{
		printf("ERROR: cuParamSetv (cudaAsyncKM2)\n");
		return;
	}
	offset += sizeof(x_numRows);

	offset = (offset + __alignof(x_numCols) - 1) & ~(__alignof(x_numCols) - 1);
	if(cuParamSeti(func, offset, x_numCols) != hipSuccess)
	{
		printf("ERROR: cuParamSetv (cudaAsyncKM2)\n");
		return;
	}
	offset += sizeof(x_numCols);

	if(cuParamSetSize(func, offset) != hipSuccess)
	{
		printf("ERROR: cuParamSetSize (cudaAsyncKM2)\n");
		return;
	}

	if(cuFuncSetBlockShape(func, x_numCols, 1, 1) != hipSuccess)
	{
		printf("ERROR: cuFuncSetBlockShape (cudaAsyncKM2)\n");
		return;
	}

    //printf("Num rows %d",x_numRows);

	if(cuLaunchGridAsync(func, 1, mu_numRows, (hipStream_t)stream) != hipSuccess)
	{
		printf("ERROR: cuLaunchGrid (cudaAsyncKM2)\n");
		return;
	}
}


JNIEXPORT void JNICALL Java_ppl_delite_cuda_DeliteCudaDriver_cudaAsyncKM3(JNIEnv *env, jobject obj, jlong func_id, jlong a_ptr, jlong b_ptr, jlong c_ptr, jint numRows, jint numCols, jlong stream)
{
	int offset = 0;
	void* ptr;
	hipFunction_t func = (hipFunction_t)func_id;

	ptr = (void*)(size_t)a_ptr;
	offset = (offset + __alignof(ptr) - 1) & ~(__alignof(ptr) - 1);
	if(cuParamSetv(func, offset, &ptr, sizeof(ptr)) != hipSuccess)
	{
		printf("ERROR: cuParamSetv (cudaAsyncKM3)\n");
		return;
	}
	offset += sizeof(ptr);

    ptr = (void*)(size_t)b_ptr;
	offset = (offset + __alignof(ptr) - 1) & ~(__alignof(ptr) - 1);
	if(cuParamSetv(func, offset, &ptr, sizeof(ptr)) != hipSuccess)
	{
		printf("ERROR: cuParamSetv (cudaAsyncKM3)\n");
		return;
	}
	offset += sizeof(ptr);

	ptr = (void*)(size_t)c_ptr;
	offset = (offset + __alignof(ptr) - 1) & ~(__alignof(ptr) - 1);
	if(cuParamSetv(func, offset, &ptr, sizeof(ptr)) != hipSuccess)
	{
		printf("ERROR: cuParamSetv (cudaAsyncKM3)\n");
		return;
	}
	offset += sizeof(ptr);

	offset = (offset + __alignof(numCols) - 1) & ~(__alignof(numCols) - 1);
	if(cuParamSeti(func, offset, numCols) != hipSuccess)
	{
		printf("ERROR: cuParamSetv (cudaAsyncKM3)\n");
		return;
	}
	offset += sizeof(numCols);

	if(cuParamSetSize(func, offset) != hipSuccess)
	{
		printf("ERROR: cuParamSetSize (cudaAsyncKM3)\n");
		return;
	}

	if(cuFuncSetBlockShape(func, 512, 1, 1) != hipSuccess)
	{
		printf("ERROR: cuFuncSetBlockShape (cudaAsyncKM3)\n");
		return;
	}

	if(cuLaunchGridAsync(func, 1+(numCols-1)/512, numRows, (hipStream_t)stream) != hipSuccess)
	{
		printf("ERROR: cuLaunchGrid (cudaAsyncKM3)\n");
		return;
	}
}


JNIEXPORT void JNICALL Java_ppl_delite_cuda_DeliteCudaDriver_cudaAsyncMdotV(JNIEnv *env, jobject obj, jlong func_id, jlong a_ptr, jlong b_ptr, jlong c_ptr, jint height, jint width, jlong stream)
{
	int offset = 0;
	void* ptr;
	hipFunction_t func = (hipFunction_t)func_id;

	ptr = (void*)(size_t)a_ptr;
	offset = (offset + __alignof(ptr) - 1) & ~(__alignof(ptr) - 1);
	if(cuParamSetv(func, offset, &ptr, sizeof(ptr)) != hipSuccess)
	{
		printf("ERROR: cuParamSetv (cudaAsync3D2I)\n");
		return;
	}
	offset += sizeof(ptr);

	ptr = (void*)(size_t)b_ptr;
	offset = (offset + __alignof(ptr) - 1) & ~(__alignof(ptr) - 1);
	if(cuParamSetv(func, offset, &ptr, sizeof(ptr)) != hipSuccess)
	{
		printf("ERROR: cuParamSetv (cudaAsync3D2I)\n");
		return;
	}
	offset += sizeof(ptr);

	ptr = (void*)(size_t)c_ptr;
	offset = (offset + __alignof(ptr) - 1) & ~(__alignof(ptr) - 1);
	if(cuParamSetv(func, offset, &ptr, sizeof(ptr)) != hipSuccess)
	{
		printf("ERROR: cuParamSetv (cudaAsync3D2I)\n");
		return;
	}
	offset += sizeof(ptr);

	offset = (offset + __alignof(width) - 1) & ~(__alignof(width) - 1);
	if(cuParamSeti(func, offset, width) != hipSuccess)
	{
		printf("ERROR: cuParamSetv (cudaAsync3D2I)\n");
		return;
	}
	offset += sizeof(width);

	offset = (offset + __alignof(height) - 1) & ~(__alignof(height) - 1);
	if(cuParamSeti(func, offset, height) != hipSuccess)
	{
		printf("ERROR: cuParamSetv (cudaAsync3D2I)\n");
		return;
	}
	offset += sizeof(height);

	if(cuParamSetSize(func, offset) != hipSuccess)
	{
		printf("ERROR: cuParamSetSize (cudaAsync3D2I)\n");
		return;
	}

	//TODO : Need to receive dimension info as inputs
	if(cuFuncSetBlockShape(func, 4, 128, 1) != hipSuccess)
	{
		printf("ERROR: cuFuncSetBlockShape (cudaAsync3D2I)\n");
		return;
	}

	if(cuLaunchGridAsync(func, 1, 1+(height-1)/128, (hipStream_t)stream) != hipSuccess)
	{
		printf("ERROR: cuLaunchGrid (cudaAsync3D2I)\n");
		return;
	}
}


JNIEXPORT void JNICALL Java_ppl_delite_cuda_DeliteCudaDriver_cudaAsyncMatDotV(JNIEnv *env, jobject obj, jlong func_id, jlong a_ptr, jlong b_ptr, jlong c_ptr, jint width, jint height, jlong stream)
{
	int offset = 0;
	void* ptr;
	hipFunction_t func = (hipFunction_t)func_id;

	ptr = (void*)(size_t)a_ptr;
	offset = (offset + __alignof(ptr) - 1) & ~(__alignof(ptr) - 1);
	if(cuParamSetv(func, offset, &ptr, sizeof(ptr)) != hipSuccess)
	{
		printf("ERROR: cuParamSetv (cudaAsync3D2I)\n");
		return;
	}
	offset += sizeof(ptr);

	ptr = (void*)(size_t)b_ptr;
	offset = (offset + __alignof(ptr) - 1) & ~(__alignof(ptr) - 1); 
	if(cuParamSetv(func, offset, &ptr, sizeof(ptr)) != hipSuccess)
	{
		printf("ERROR: cuParamSetv (cudaAsync3D2I)\n");
		return;
	}
	offset += sizeof(ptr);
	
	ptr = (void*)(size_t)c_ptr;
	offset = (offset + __alignof(ptr) - 1) & ~(__alignof(ptr) - 1); 
	if(cuParamSetv(func, offset, &ptr, sizeof(ptr)) != hipSuccess)
	{
		printf("ERROR: cuParamSetv (cudaAsync3D2I)\n");
		return;
	}
	offset += sizeof(ptr);

	offset = (offset + __alignof(width) - 1) & ~(__alignof(width) - 1);
	if(cuParamSeti(func, offset, width) != hipSuccess)
	{
		printf("ERROR: cuParamSetv (cudaAsync3D2I)\n");
		return;
	}
	offset += sizeof(width);
	
	offset = (offset + __alignof(height) - 1) & ~(__alignof(height) - 1);
	if(cuParamSeti(func, offset, height) != hipSuccess)
	{
		printf("ERROR: cuParamSetv (cudaAsync3D2I)\n");
		return;
	}
	offset += sizeof(height);
	
	if(cuParamSetSize(func, offset) != hipSuccess)
	{
		printf("ERROR: cuParamSetSize (cudaAsync3D2I)\n");
		return;
	}


	//TODO : Need to receive dimension info as inputs
	if(cuFuncSetBlockShape(func, 256, 1, 1) != hipSuccess)
	{
		printf("ERROR: cuFuncSetBlockShape (cudaAsync3D2I)\n");
		return;
	}

	if(cuLaunchGridAsync(func, 1+(width-1)/256, height, (hipStream_t)stream) != hipSuccess)
	{
		printf("ERROR: cuLaunchGrid (cudaAsync3D2I)\n");
		return;
	}
}

JNIEXPORT void JNICALL Java_ppl_delite_cuda_DeliteCudaDriver_cudaAsync3D3IReg(JNIEnv *env, jobject obj, jlong func_id, jlong a_ptr, jlong b_ptr, jlong c_ptr, jint widthA, jint widthB, jint heightA, jlong stream)
{
	int offset = 0;
	void* ptr;
	hipFunction_t func = (hipFunction_t)func_id;

	ptr = (void*)(size_t)a_ptr;
	offset = (offset + __alignof(ptr) - 1) & ~(__alignof(ptr) - 1);
	if(cuParamSetv(func, offset, &ptr, sizeof(ptr)) != hipSuccess)
	{
		printf("ERROR: cuParamSetv (cudaAsync3D3I)\n");
		return;
	}
	offset += sizeof(ptr);

	ptr = (void*)(size_t)b_ptr;
	offset = (offset + __alignof(ptr) - 1) & ~(__alignof(ptr) - 1); 
	if(cuParamSetv(func, offset, &ptr, sizeof(ptr)) != hipSuccess)
	{
		printf("ERROR: cuParamSetv (cudaAsync3D3I)\n");
		return;
	}
	offset += sizeof(ptr);
	
	ptr = (void*)(size_t)c_ptr;
	offset = (offset + __alignof(ptr) - 1) & ~(__alignof(ptr) - 1); 
	if(cuParamSetv(func, offset, &ptr, sizeof(ptr)) != hipSuccess)
	{
		printf("ERROR: cuParamSetv (cudaAsync3D3I)\n");
		return;
	}
	offset += sizeof(ptr);

	offset = (offset + __alignof(widthA) - 1) & ~(__alignof(widthA) - 1);
	if(cuParamSeti(func, offset, widthA) != hipSuccess)
	{
		printf("ERROR: cuParamSetv (cudaAsync3D3I)\n");
		return;
	}
	offset += sizeof(widthA);
	
	offset = (offset + __alignof(widthB) - 1) & ~(__alignof(widthB) - 1);
	if(cuParamSeti(func, offset,  widthB) != hipSuccess)
	{
		printf("ERROR: cuParamSetv (cudaAsync3D3I)\n");
		return;
	}
	offset += sizeof(widthB);
	
	offset = (offset + __alignof(heightA) - 1) & ~(__alignof(heightA) - 1);
	if(cuParamSeti(func, offset, heightA) != hipSuccess)
	{
		printf("ERROR: cuParamSetv (cudaAsync3D3I)\n");
		return;
	}
	offset += sizeof(heightA);

	
	if(cuParamSetSize(func, offset) != hipSuccess)
	{
		printf("ERROR: cuParamSetSize (cudaAsync3D3I)\n");
		return;
	}


	//TODO : Need to receive dimension info as inputs
	if(cuFuncSetBlockShape(func, 16, 16, 1) != hipSuccess)
	{
		printf("ERROR: cuFuncSetBlockShape (cudaAsync3D3I)\n");
		return;
	}

	if(widthB == 0) {
	    if(cuLaunchGridAsync(func, 1+(widthA-1)/16, 1+(heightA-1)/16, (hipStream_t)stream) != hipSuccess)
	    {
		    printf("ERROR: cuLaunchGrid (cudaAsync3D3I)\n");
		    return;
	    }
	}
	else {
        if(cuLaunchGridAsync(func, 1+(widthB-1)/16, 1+(heightA-1)/16, (hipStream_t)stream) != hipSuccess)
	    {
		    printf("ERROR: cuLaunchGrid (cudaAsync3D3I)\n");
		    return;
	    }
	}
}
*/	
