#include <stdio.h>
#include "hip/hip_runtime.h"
#include "ppl_delite_cuda_DeliteCudaDriver.h"

JNIEXPORT void JNICALL Java_ppl_delite_cuda_DeliteCudaDriver_dummy(JNIEnv *env, jobject obj)
{
	return;
}

JNIEXPORT void JNICALL Java_ppl_delite_cuda_DeliteCudaDriver_cudaInit(JNIEnv *env, jclass cls)
{
    if(hipInit(0) != hipSuccess)
    {
         printf("ERROR: hipInit\n");
    }
}


JNIEXPORT jlong JNICALL Java_ppl_delite_cuda_DeliteCudaDriver_cudaDevSet(JNIEnv *env, jobject obj, jint devIdx)
{
	hipDevice_t dev;
	hipCtx_t pctx;
	hipModule_t module;

    /*
    int deviceNum;
    if(hipGetDeviceCount(&deviceNum) != hipSuccess) {
        printf("ERROR: hipGetDeviceCount\n");
            return -1;
    }
    printf("Initializing %d device out of %d devices\n", devIdx, deviceNum);
    */
    
    if(hipDeviceGet(&dev, devIdx) != hipSuccess)
    {
        printf("ERROR: hipDeviceGet\n");
        return -1;
    }

    if(hipCtxCreate(&pctx, hipDeviceMapHost, dev) != hipSuccess)
    {
        printf("ERROR: hipCtxCreate\n");
        return -1;
    }

    if(hipModuleLoad(&module, "/tmp/cuda/cudaKernels.ptx") != hipSuccess)
    {
        printf("ERROR: hipModuleLoad\n");
        return -1;
    }
	return (jlong)(module);
}


JNIEXPORT void JNICALL Java_ppl_delite_cuda_DeliteCudaDriver_cudaDestroyContext(JNIEnv *env, jobject obj, jint devIdx)
{
	hipCtx_t pctx;
    if(hipCtxPopCurrent(&pctx) != hipSuccess) {
        printf("ERROR: hipCtxPopCurrent\n");
        return;
    }

    if(hipCtxDestroy(pctx) != hipSuccess) {
        printf("ERROR: hipCtxDestroy\n");
        return;
    }
}

JNIEXPORT jlong JNICALL Java_ppl_delite_cuda_DeliteCudaDriver_cudaCreateStream(JNIEnv *env, jobject obj)
{
	hipStream_t stream;

	if(hipStreamCreateWithFlags(&stream, 0) != hipSuccess)
	{
		printf("ERROR: hipModuleGetFunction\n");
		return -1;
	}

	return (jlong)stream;
}

JNIEXPORT void JNICALL Java_ppl_delite_cuda_DeliteCudaDriver_cudaDestroyStream(JNIEnv *env, jobject obj, jlong stream)
{
	if(hipStreamDestroy((hipStream_t)stream) != hipSuccess)
	{
		printf("ERROR: hipStreamDestroy\n");
		return;
	}
}

JNIEXPORT jlong JNICALL Java_ppl_delite_cuda_DeliteCudaDriver_cudaGetFunction(JNIEnv *env, jobject obj, jlong module, jstring kernel)
{
	hipFunction_t func;
	
	// Get Kernel name character pointer
	const char *KernelName = env->GetStringUTFChars(kernel, 0);

	if(hipModuleGetFunction(&func, (hipModule_t)module, (const char *)KernelName) != hipSuccess)
	{
		printf("ERROR: hipModuleGetFunction: %s\n", KernelName);
		return -1;
	}

	// Let Java VM know that the char pointer is not needed by the native code
	env->ReleaseStringUTFChars(kernel, KernelName);

	return (jlong)func;

}

JNIEXPORT jint JNICALL Java_ppl_delite_cuda_DeliteCudaDriver_cudaGetDevNum(JNIEnv *env, jobject obj)
{
	jint dev_num = 0;

	hipGetDeviceCount(&dev_num);
	
	return dev_num;
}

JNIEXPORT void JNICALL Java_ppl_delite_cuda_DeliteCudaDriver_cudaStreamSync(JNIEnv *env, jobject obj, jlong stream)
{
	if(hipStreamSynchronize((hipStream_t)stream) != hipSuccess)
	{
		printf("ERROR: cudaStreamSync\n");
		return;
	}
	if(hipStreamQuery((hipStream_t)stream) != hipSuccess)
	{
		printf("ERROR: cudaStreamQeury\n");
		return;
	}
}

