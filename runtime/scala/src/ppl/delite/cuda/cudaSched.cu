#include <stdio.h>
#include "hip/hip_runtime.h"
#include "ppl_delite_cuda_DeliteCudaDriver.h"

JNIEXPORT void JNICALL Java_ppl_delite_cuda_DeliteCudaDriver_cudaSched(JNIEnv *env, jobject obj, jlong func_id, jlong cnt_ptr, jint count, jlong stream)
{
	int offset = 0;
	void* ptr;
	hipFunction_t func = (hipFunction_t)func_id;

	ptr = (void*)(size_t)cnt_ptr;
	offset = (offset + __alignof(ptr) - 1) & ~(__alignof(ptr) - 1);
	if(cuParamSetv(func, offset, &ptr, sizeof(ptr)) != hipSuccess)
	{
		printf("ERROR: cuParamSetv (cudaSched)\n");
		return;
	}
	offset += sizeof(ptr);

	offset = (offset + __alignof(count) - 1) & ~(__alignof(count) - 1);
	if(cuParamSeti(func, offset, count) != hipSuccess)
	{
		printf("ERROR: cuParamSetv (cudaSched)\n");
		return;
	}
	offset += sizeof(count);

	if(cuParamSetSize(func, offset) != hipSuccess)
	{
		printf("ERROR: cuParamSetSize (cudaSched)\n");
		return;
	}
	
	if(cuFuncSetBlockShape(func, 1, 1, 1) != hipSuccess)
	{
		printf("ERROR: cuFuncSetBlockShape (cudaSched)\n");
		return;
	}

	if(cuLaunchGridAsync(func, 1, 1, (hipStream_t)stream) != hipSuccess)
	{
		printf("ERROR: cuLaunchGrid (cudaSched)\n");
		return;
	}
	
}

JNIEXPORT jint JNICALL Java_ppl_delite_cuda_DeliteCudaDriver_readDevTimeStamp(JNIEnv *env, jobject obj, jlong TS_ptr)
{
	int *ptr = (int *)TS_ptr;
	return (*ptr);
}

