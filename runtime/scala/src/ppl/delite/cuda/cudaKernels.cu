#include "hip/hip_runtime.h"
#include "matMultDouble.cu"
#include "matMultFloat.cu"
#include "matrixMul_kernel.cu"

__global__ void vectPlusDouble(double *A, double *B, double *C, int size)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	
	while (i < size)
	{
		C[i] = A[i] + B[i];
		i += blockDim.x * gridDim.x;
	}

	__syncthreads();
}

__global__ void vectPlusFloat(float *A, float *B, float *C, int size)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	
	while (i < size)
	{
		C[i] = A[i] + B[i];
		i += blockDim.x * gridDim.x;
	}

	__syncthreads();
}

__global__ void vectMinusDouble(double *A, double *B, double *C, int size)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	while (i < size)
	{
		C[i] = A[i] - B[i];
		i += blockDim.x * gridDim.x;
	}

	__syncthreads();
}

__global__ void vectMinusFloat(float *A, float *B, float *C, int size)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	while (i < size)
	{
		C[i] = A[i] - B[i];
		i += blockDim.x * gridDim.x;
	}

	__syncthreads();
}

__global__ void vectMultDouble(double *A, double *B, double *C, int size)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	while (i < size)
	{
		C[i] = A[i] * B[i];
		i += blockDim.x * gridDim.x;
	}

	__syncthreads();
}

__global__ void vectMultFloat(float *A, float *B, float *C, int size)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	while (i < size)
	{
		C[i] = A[i] * B[i];
		i += blockDim.x * gridDim.x;
	}

	__syncthreads();
}

__global__ void vectMultInt(int *A, int *B, int *C, int size)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	while (i < size)
	{
		C[i] = A[i] * B[i];
		i += blockDim.x * gridDim.x;
	}

	__syncthreads();
}

__global__ void vectDivDouble(double *A, double *B, double *C, int size)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	while (i < size)
	{
		C[i] = A[i] / B[i];
		i += blockDim.x * gridDim.x;
	}

	__syncthreads();
}

__global__ void vectDivFloat(float *A, float *B, float *C, int size)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	while (i < size)
	{
		C[i] = A[i] / B[i];
		i += blockDim.x * gridDim.x;
	}

	__syncthreads();
}

__global__ void vectMoveDouble(double *A, double *C, int size)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	
	while (i < size)
	{
		C[i] = A[i];
		i += blockDim.x * gridDim.x;
	}

	__syncthreads();
}

__global__ void vectMoveFloat(float *A, float *C, int size)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	
	while (i < size)
	{
		C[i] = A[i];
		i += blockDim.x * gridDim.x;
	}

	__syncthreads();
}

__global__ void vectMoveInt(int *A, int *B, int *C, int size)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	while (i < size)
	{
		C[i] = A[i];
		i += blockDim.x * gridDim.x;
	}

	__syncthreads();
}

__global__ void vectGTDouble(double *A, double *B, double *C, int size)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i < size) {
	    if(A[i] > B[i])
	        C[i] = 1;
	    else
	        C[i] = 0;
	}

	__syncthreads();
}

__global__ void vectGTFloat(float *A, float *B, float *C, int size)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i < size) {
	    if(A[i] > B[i])
	        C[i] = 1;
	    else
	        C[i] = 0;
	}

	__syncthreads();
}

__global__ void vectEQDouble(double *A, double *B, double *C, int size)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i < size) {
	    if(A[i] == B[i])
	        C[i] = 1;
	    else
	        C[i] = 0;
	}

	__syncthreads();
}

__global__ void vectEQFloat(float *A, float *B, float *C, int size)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i < size) {
	    if(A[i] == B[i])
	        C[i] = 1;
	    else
	        C[i] = 0;
	}

	__syncthreads();
}

__global__ void vectRecipDouble(double *A, double *C, int size)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i < size)
	    C[i] = 1 / A[i];

	__syncthreads();
}

__global__ void vectRecipFloat(float *A, float *C, int size)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i < size)
	    C[i] = 1 / A[i];

	__syncthreads();
}

__global__ void vectExpDouble(double *A, double *C, int size)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i < size)
	    C[i] = exp(A[i]);

	__syncthreads();
}

__global__ void vectExpFloat(float *A, float *C, int size)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i < size)
	    C[i] = exp(A[i]);

	__syncthreads();
}


__global__ void matOuterDouble(double *A, double *B, double *C, int size)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	int j = 0;

	__shared__ double S_A[16];
	__shared__ double S_B[16];

	S_A[threadIdx.x] = A[i];
	S_B[threadIdx.x] = B[blockIdx.y*16+threadIdx.x];

	__syncthreads();

	for(j=0; j<16; j++) {
		C[i+(j+blockIdx.y*16)*size] = S_A[threadIdx.x] * S_B[j];
	}

	__syncthreads();
}

__global__ void matOuterFloat(float *A, float *B, float *C, int size)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	int j = 0;

	__shared__ float S_A[16];
	__shared__ float S_B[16];

	S_A[threadIdx.x] = A[i];
	S_B[threadIdx.x] = B[blockIdx.y*16+threadIdx.x];
	
	__syncthreads();

	for(j=0; j<16; j++) {
		C[i+(j+blockIdx.y*16)*size] = S_A[threadIdx.x] * S_B[j];
	}

	__syncthreads();
}


/* 
 * preKernel 
 * Wait until(busy waiting) the counter(epoch) becomes desired value
 */
__global__ void preKernel(int *cntPtr, int cnt)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	
	if(i == 0) {
		while(*cntPtr < cnt) {*(cntPtr+1) = 0;}
		//while(*cntPtr < cnt) {}
	}
	__syncthreads();
}

/* 
 * postKernel 
 * Set the counter to desired counter value
 */
__global__ void postKernel(int *cntPtr, int cnt)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	
	if(i == 0) {
		*cntPtr = cnt;
	}
	__syncthreads();
}

__global__ void mapNB(double *features, double *classifications, double *phi, int width, int height, int compareWith, double weightedspamcount) 
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	int j = 0;

	double spamwordcount = 0.0;
	for(j=0; j<height; j++) {
		if(classifications[j] == compareWith)
			spamwordcount = spamwordcount + features[j*width+i];
	}

	if(i < width)
		phi[i] = (spamwordcount + 1) / (weightedspamcount + width);

	__syncthreads();
	
	/*
	int j = blockDim.x * blockIdx.x + threadIdx.x;
	int i = 0;

	__shared__ double spamwordcount[512];
	double value;

	spamwordcount[threadIdx.y][threadIdx.x] = 0;

	while(i*16 + threadIdx.y < height) {
		value = features[(i*16+threadIdx.y)*width+j];
		if(classifications[i*16+threadIdx.y] == compareWith)
			spamwordcount[threadIdx.y][threadIdx.x] += value;
		i += 1;
	}
	__syncthreads();

	if(threadIdx.y == 0) {
		for(i=1; i<16; i++)
			spamwordcount[0][threadIdx.x] += spamwordcount[i][threadIdx.x];
		if(j < width)
			phi[j] = (spamwordcount[0][threadIdx.x] + 1) / (weightedspamcount + width);
	}
	__syncthreads();
	*/

}

__global__ void mapToVecNB(double *matrix, double *out, int width, int height) 
{
/*
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	int j = 0;

	__shared__ double localsum[128];

	localsum[threadIdx.x] = 0;

    if(i < height) {
	    while(j < width) {
		    localsum[threadIdx.x] += matrix[i*width+j];
		    j += 1;
		}
		out[i] = localsum[threadIdx.x];
	}
	__syncthreads();
*/
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	int j = 0;

	__shared__ double localsum[256];

	localsum[threadIdx.x] = 0;

    if(i < width) {
	    while(j < height) {
		    localsum[threadIdx.x] += matrix[j*width+i];
		    j += 1;
		}
		out[i] = localsum[threadIdx.x];
	}
	__syncthreads();

}

__global__ void mapLR(double *input, double *output, double x_cur, int tau, int count) 
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	double base = x_cur - input[i];
	
	if(i < count) {
		output[i] = exp((-0.1*base*base)/(2.0*tau*tau));
	}
	__syncthreads();
}

/*
__global__ void mapLRBig(double *xref, int xref_height, double *x, double *O, double *output, double x_cur, int tau, int count) 
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	
	double x_cur = xref[i*xref_height];
	if(i < count) {
		output[i] = exp((-0.1*base*base)/(2.0*tau*tau));
	}
	__syncthreads();
}
*/

__global__ void dotVMDouble(double *inputM, double *inputV, double *outputM, int width, int height) 
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	int j = blockDim.y * blockIdx.y + threadIdx.y;
	//int h = 0;
	//__shared__ double v;

	//while(h < height) {
		//if(threadIdx.x == 0)
		//	v = inputV[j];
		//__syncthreads();
		if(i < width)
			outputM[j*width+i] = inputM[j*width+i] * inputV[j];
	//	h += 1;
	//}
	__syncthreads();
}

__global__ void matTransDouble(double *input, double *output, int width, int height)
{
    int j = blockDim.x * blockIdx.x + threadIdx.x;
    int i = blockDim.y * blockIdx.y + threadIdx.y;

    __shared__ double S_input[16][16];

    // Read inputs to shared mem
    if((j<width) && (i<height))
        S_input[threadIdx.y][threadIdx.x] = input[i*width+j];
    __syncthreads();

    // Write result to device mem
    int jj = blockDim.y * blockIdx.y + threadIdx.x;
    int ii = blockDim.x * blockIdx.x + threadIdx.y;
    if((jj<height) && (ii<width))
        output[ii*height+jj] = S_input[threadIdx.x][threadIdx.y];

    __syncthreads();

}

__global__ void matTransFloat(float *input, float *output, int width, int height)
{
    int j = blockDim.x * blockIdx.x + threadIdx.x;
    int i = blockDim.y * blockIdx.y + threadIdx.y;

    __shared__ float S_input[16][16];

    // Read inputs to shared mem
    if((j<width) && (i<height))
        S_input[threadIdx.y][threadIdx.x] = input[i*width+j];
    __syncthreads();

    // Write result to device mem
    int jj = blockDim.y * blockIdx.y + threadIdx.x;
    int ii = blockDim.x * blockIdx.x + threadIdx.y;
    if((jj<height) && (ii<width))
        output[ii*height+jj] = S_input[threadIdx.x][threadIdx.y];

    __syncthreads();

}

__global__ void matInvDouble(double *input, double *output, int dum1, int dum2) 
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	__shared__ double arr[4][4];

	arr[0][i] = input[i];
	arr[1][i] = input[i];
	arr[2][i] = input[i];
	arr[3][i] = input[i];

	double admbc = arr[i][0]*arr[i][3] - arr[i][1]*arr[i][2];

    if(i < 4)
	    output[i] = arr[i][3-i]/admbc;

	if((i==1)||(i==2))
		output[i] *= -1;
	
	__syncthreads();
}

#define DIM_X 16
#define DIM_Y 32

__global__ void MprodVDouble(double *inputM, double *inputV, double *outputV, int width, int height) 
{
	/*
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	int j = blockDim.y * blockIdx.y + threadIdx.y;
    int iter = (511+width) / 512;
    int k = 0;
    int step = 0;
    __shared__ double tempM[512];
    __shared__ double tempV[512];
    __shared__ double localSum[512];
    double finalSum = 0.0;
    
    //get inputs
    while(k < iter) {
        tempM[threadIdx.x] = 0.0;
        tempV[threadIdx.x] = 0.0;

        if(i < width) {
            tempM[threadIdx.x] = inputM[j*width+i];
            tempV[threadIdx.x] = inputV[i];
        }
        localSum[threadIdx.x] = tempM[threadIdx.x] * tempV[threadIdx.x];
        step = 1;
        __syncthreads();

        // Do reduction
        while(step < 512) {
            if(threadIdx.x % (2*step) == 0)
                localSum[threadIdx.x] += localSum[threadIdx.x+step];
            __syncthreads();
            step *= 2;
        }
        if(threadIdx.x == 0)
            finalSum += localSum[0];
        k += 1;
        i += 512;
    }
    
    if(threadIdx.x == 0)
        outputV[j] = finalSum;
    */

	//int i = blockDim.x * blockIdx.x + threadIdx.x;
	int j = blockDim.y * blockIdx.y + threadIdx.y;
	double localsum = 0;
    int ii = 0;
    int iii = 0;
    
	__shared__ double temp[DIM_Y][DIM_X];

	temp[threadIdx.y][threadIdx.x] = 0.0;

	while(ii+DIM_X < width) {
	    temp[threadIdx.y][threadIdx.x] += inputV[ii+threadIdx.x] * inputM[j*width+ii+threadIdx.x];
		/*
	    __syncthreads();
	    if(threadIdx.x == 0) {
	        iii = 0;
	        while(iii < DIM_X) {
	            localsum += temp[threadIdx.y][iii];
	            iii += 1;
	        }
	    }
		*/
	    ii += DIM_X;
	}
	__syncthreads();
	if(threadIdx.x == 0) {
	    iii = 0;
	    while(iii < DIM_X) {
	        localsum += temp[threadIdx.y][iii];
	        iii += 1;
	    }
	}

    if(threadIdx.x == 0) {
        while(ii < width) {
            localsum += (inputV[ii]*inputM[j*width+ii]);
            ii += 1;
        }
        if(j < height)
            outputV[j] = localsum;
    }
    __syncthreads();
    

        
/*
    while(h < width) {
        localsum += (inputV[h]*inputM[i*width+h]);
        h += 1;
    }
    if(i < height)
        outputV[i] = localsum;
*/

	__syncthreads();
}

__global__ void MprodVInt(int *inputM, int *inputV, int *outputV, int width, int height) 
{
	//int i = blockDim.x * blockIdx.x + threadIdx.x;
	int j = blockDim.y * blockIdx.y + threadIdx.y;

	double localsum = 0;
    int ii = 0;
    int iii = 0;

	__shared__ int temp[DIM_Y][DIM_X];

	while(ii+DIM_X < width) {
	    temp[threadIdx.y][threadIdx.x] = inputV[ii+threadIdx.x] * inputM[j*width+ii+threadIdx.x];
	    __syncthreads();

	    if(threadIdx.x == 0) {
	        iii = 0;
	        while(iii < DIM_X) {
	            localsum += temp[threadIdx.y][iii];
	            iii += 1;
	        }
	    }
	    ii += DIM_X;
	}

    if(threadIdx.x == 0) {
        while(ii < width) {
            localsum += (inputV[ii]*inputM[j*width+ii]);
            ii += 1;
        }
        if(j < height)
            outputV[j] = localsum;
    }
    __syncthreads();

	/*
	//int i = blockDim.x * blockIdx.x + threadIdx.x;
	int indx = 0;
	int h = 0;
	int stride = 2;
	int iter = 0;
	int mValue;
	int vValue;
	int result = 0;
	//__shared__ double arrV[256];
	__shared__ int resV[256];

	//arrV[threadIdx.x] = 0;
	resV[threadIdx.x] = 0;

	while(h < height) {
	    result = 0;
		iter = 0;
		//indx = blockDim.x*blockIdx.x+threadIdx.x;

		while(iter < 1+(width-1)/256) {
			//arrV[i] = inputV[indx];
			indx = iter * 256 + threadIdx.x;
			resV[threadIdx.x] = 0;
			mValue = 0;
			vValue = 0;
			if(indx < width) {
				vValue = inputV[indx];
				mValue = inputM[indx+h*width];
			}
			resV[threadIdx.x] = mValue * vValue;

			__syncthreads();
			stride = 1;
			while(stride <= 128) {
				if(threadIdx.x % (stride*2) == 0)
					resV[threadIdx.x] += resV[threadIdx.x+stride];
				__syncthreads();
				stride *= 2;
			}
			//__syncthreads();
			if(threadIdx.x==0)
				result += resV[threadIdx.x];
			iter += 1;
			//__syncthreads();
		}
		if(threadIdx.x==0)
			outputV[h] = result;
		//__syncthreads();
		h += 1;
	}
	*/
	__syncthreads();
}

/* Kernels for RBM */
__global__ void vectPlusDouble_S(double *A, double *C, double B, int size)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i < size)
		C[i] = A[i] + B;

	__syncthreads();
}

__global__ void vectPlusFloat_S(float *A, float *C, float B, int size)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i < size)
		C[i] = A[i] + B;

	__syncthreads();
}

__global__ void vectMinusDouble_S(double *A, double *C, double B, int size)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i < size)
		C[i] = A[i] - B;

	__syncthreads();
}

__global__ void vectMinusFloat_S(float *A, float *C, float B, int size)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i < size)
		C[i] = A[i] - B;

	__syncthreads();
}

__global__ void vectMultDouble_S(double *A, double *C, double B, int size)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i < size)
		C[i] = A[i] * B;

	__syncthreads();
}

__global__ void vectMultFloat_S(float *A, float *C, float B, int size)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i < size)
		C[i] = A[i] * B;

	__syncthreads();
}

__global__ void vectDivDouble_S(double *A, double *C, double B, int size)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i < size)
		C[i] = A[i] / B;

	__syncthreads();
}

__global__ void vectDivFloat_S(float *A, float *C, float B, int size)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i < size)
		C[i] = A[i] / B;

	__syncthreads();
}

__global__ void vectRepDouble(double *A, double *C, int size, int repRow, int repCol)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
    int j = blockDim.y * blockIdx.y + threadIdx.y; //(threadIdx.y==0 always

	if (i < size*repCol)
		C[j*size*repCol+i] = A[i%size];

	__syncthreads();
}

__global__ void vectRepFloat(float *A, float *C, int size, int repRow, int repCol)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
    int j = blockDim.y * blockIdx.y + threadIdx.y; //(threadIdx.y==0 always

	if (i < size*repCol)
		C[j*size*repCol+i] = A[i%size];

	__syncthreads();
}

__global__ void sumColsDouble(double *A, double *C, int numRows, int numCols)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
    int j = 0;

    __shared__ double localSum[512];
    localSum[threadIdx.x] = 0;
    
    while(j < numRows) {
        if(i < numCols)
            localSum[threadIdx.x] += A[j*numCols+i];
        j += 1;
    }
    
    if(i < numCols)
        C[i] = localSum[threadIdx.x];
                  
	__syncthreads();
}

__global__ void sumColsFloat(float *A, float *C, int numRows, int numCols)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
    int j = 0;

    __shared__ float localSum[512];
    localSum[threadIdx.x] = 0;

    while(j < numRows) {
        if(i < numCols)
            localSum[threadIdx.x] += A[j*numCols+i];
        j += 1;
    }

    if(i < numCols)
        C[i] = localSum[threadIdx.x];

	__syncthreads();
}

__global__ void sumColsPredDouble(double *A, double *pred, double *C, int numRows, int numCols)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
    int j = 0;

    __shared__ double localSum[512];
    localSum[threadIdx.x] = 0;

    while(j < numRows) {
        if(i < numCols) {
            if(pred[j] == 1.0)
                localSum[threadIdx.x] += A[j*numCols+i];
        }
        j += 1;
    }

    if(i < numCols)
        C[i] = localSum[threadIdx.x];

	__syncthreads();
}

__global__ void sumColsPredFloat(float *A, float *pred, float *C, int numRows, int numCols)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
    int j = 0;

    __shared__ float localSum[512];
    localSum[threadIdx.x] = 0;

    while(j < numRows) {
        if(i < numCols) {
            if(pred[j] == 1.0)
                localSum[threadIdx.x] += A[j*numCols+i];
        }
        j += 1;
    }

    if(i < numCols)
        C[i] = localSum[threadIdx.x];

	__syncthreads();
}

//Input: 2 double matrices (x, mu)
//Output: 1 double vector (out)
__global__ void mapKM1(double *x, double *mu, int *out, int x_numRows, int x_numCols, int mu_numRows, int mu_numCols)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	int j,k;
	double min_d = 100000;
	int min_j = -1;
	double dist = 0.0;
	double diff;
	
	if(i < x_numRows) {
		for(j=0; j<mu_numRows; j++) {
			dist = 0.0;
			for(k=0; k<x_numCols; k++) {
				diff = x[i*x_numCols+k] - mu[j*mu_numCols+k];
				dist += diff * diff;
			}
			if(dist < min_d) {
				min_d = dist;
				min_j = j;
			}
		}
		out[i] = min_j;
	}

/*
	
    // Sequential Code (only threadIdx==0 working)
    double diff = 0.0;
    double sum = 0.0;
    int l = 0;
    if(i==0) {
        for(l=0; l<mu_numRows; l++) {
            sum = 0.0;
            for(k=0; k<x_numCols; k++) {
                diff = x[j*x_numCols+k] - mu[l*mu_numCols+k];
                sum += (diff * diff);
            }
            if(sum < min) {
                min = sum;
                min_j = l;
            }
        }
        out[j] = min_j;
    }
  */

	/*
	int i = blockDim.x * blockIdx.x + threadIdx.x;
    int j = blockDim.y * blockIdx.y + threadIdx.y;
    int k = 0;
    double min = 100000.0;     // Need to be changed to the highest value it can have
    int min_k = 0;

    int reduc_phase = 1;
    int pow2 = 1;
    __shared__ double diff[512];
    __shared__ double x_S[512];

    // Read x values
    if(i < x_numCols)
        x_S[threadIdx.x+threadIdx.y*x_numCols] = x[j*x_numCols+i];

    // calculate max number of power of 2 less than x_numCols (==mu_numCols)
    while(2*pow2 <= x_numCols)
        pow2 *= 2;
            
    for(k=0; k<mu_numRows; k++) {

        // this if-statement is not needed if the number of threads per block is exactly same as x_numCols
        if(i < x_numCols) {
            // element-wise power of 2
            diff[threadIdx.x+threadIdx.y*x_numCols] = x_S[threadIdx.x+threadIdx.y*x_numCols] - mu[k*mu_numCols+i];
            diff[threadIdx.x+threadIdx.y*x_numCols] *= diff[threadIdx.x+threadIdx.y*x_numCols];
            __syncthreads();

            // fold the array to have length of power of 2
            if(i >= pow2)
                diff[threadIdx.y*x_numCols+i-pow2] += diff[threadIdx.y*x_numCols+i];
            __syncthreads();
            
            // reduction phase (may cause problem for accessing outside the boundary of shared variable)
            reduc_phase = 1;
            while(reduc_phase != pow2) {
                if(i%(2*reduc_phase) == 0)
                    diff[threadIdx.y*x_numCols+i] += diff[threadIdx.y*x_numCols+i+reduc_phase];
                reduc_phase *= 2;
                __syncthreads();
            }

            // update minimum value & index
            if(diff[threadIdx.y*x_numCols+i] < min) {
                min = diff[threadIdx.y*x_numCols+i];
                min_k = k;
            }
        }
    }

    if(i==0) {
        out[j] = min_k;
    }
	*/

	__syncthreads();
}

__global__ void mapKM2(double *x, double *mu, int *c, int x_numRows, int x_numCols, int m_numRows)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	int j = blockDim.y * blockIdx.y + threadIdx.y;
	int k;
	int cval;
    //int points = 0;
    
	__shared__ double weightedpoints[512];
	__shared__ int points[512];

	if(i < m_numRows) {
    	//initialize weightedpoints to 0
		weightedpoints[threadIdx.y*blockDim.x+threadIdx.x] = 0.0;
		points[threadIdx.y*blockDim.x+threadIdx.x] = 0;

		for(k=threadIdx.y*blockDim.x+threadIdx.x; k<x_numRows*x_numCols; k+=(blockDim.x*blockDim.y)) {
			cval = c[k/x_numCols];
			if(cval == i) {
				weightedpoints[threadIdx.y*blockDim.x+threadIdx.x] += x[k];
				points[threadIdx.y*blockDim.x+threadIdx.x] += 1;
			}
		}
		__syncthreads();

		//Do reduction
		if(threadIdx.y < x_numCols) {
			for(k=threadIdx.y; k<blockDim.x*blockDim.y-x_numCols; k+=x_numCols) {
				weightedpoints[threadIdx.y] += weightedpoints[k+x_numCols];
				points[threadIdx.y] += points[k+x_numCols];
			}

			//Answers are in weightedpoitns[k] (k<x_numCols)
			if(points == 0) 
				mu[i*x_numCols+threadIdx.y] = 0.0;
			else
				mu[i*x_numCols+threadIdx.y] = weightedpoints[threadIdx.y] / points[threadIdx.y];
		}
	}

		/*
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	int j,k;
	int cval;
    int points = 0;
    double weightedpoints[512];

	if(i < 16) {
    	//initialize weightedpoints to 0
		for(k=0; k<x_numCols; k++)
			weightedpoints[k] = 0.0;

		for(j=0; j<x_numRows; j++) {
			cval = c[j];
			if(cval == i) {
				for(k=0; k<x_numCols; k++)
					weightedpoints[k] += x[j*x_numCols+k];
				points += 1;
			}
		}

		if(points == 0) {
			for(k=0; k<x_numCols; k++) 
				mu[i*x_numCols+k] = 0.0;
		}
		else {
			for(k=0; k<x_numCols; k++) 
				mu[i*x_numCols+k] = weightedpoints[k] / points; 
		}
	}
	*/
	__syncthreads();
}

/*
__global__ void mapKM2(double *x, double *mu, int *c, int x_numRows, int x_numCols)
{
	//int i = blockDim.x * blockIdx.x + threadIdx.x;
    int j = blockDim.y * blockIdx.y + threadIdx.y;
    int points = 0;
    int cval = 0;
    int k = 0;
    
    __shared__ double weightedpoints[512];

    //initialize weightedpoints to 0

    if(threadIdx.x < x_numCols) {
        weightedpoints[threadIdx.x] = 0.0;
        for(k=0; k<x_numRows; k++) {
            cval = c[k];
            if(cval == j) {
                weightedpoints[threadIdx.x] = weightedpoints[threadIdx.x] + x[k*x_numCols+threadIdx.x];
                points += 1;
            }
        }
        if(points == 0)
            weightedpoints[threadIdx.x] = 0.0;
        else
            mu[j*x_numCols+threadIdx.x] = weightedpoints[threadIdx.x] / points;
        //mu[j*x_numCols+threadIdx.x] = 0;
    }
	__syncthreads();
}
*/

__global__ void reduction(double *x, double *dummy, double *out, int size)
{
	//int i = blockDim.x * blockIdx.x + threadIdx.x;
	int j = 0;
    double sum = 0.0;
    
    //__shared__ double weightedpoints[512];
    if(threadIdx.x == 0) {
        for(j=0; j<size; j++)
            sum += x[j];
        out[0] = sum;
    }
    //out[0] = 2.0;

	__syncthreads();
}

__global__ void VminusMDouble(double *v, double *m, double *out, int size)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	int j = blockDim.y * blockIdx.y + threadIdx.y;

    if(i < size) {
        out[j*size+i] = v[i] - m[j*size+i];
    }
	__syncthreads();
}


__global__ void matDiagFloat(float *v, float *out, int size)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	int j = blockDim.y * blockIdx.y + threadIdx.y;

    if(i < size) {
        if(i == j)
            out[j*size+i] = v[i];
        else
            out[j*size+i] = 0;
    }
	__syncthreads();
}

__global__ void matDiagDouble(double *v, double *out, int size)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	int j = blockDim.y * blockIdx.y + threadIdx.y;

    if(i < size) {
        if(i == j)
            out[j*size+i] = v[i];
        else
            out[j*size+i] = 0;
    }
	__syncthreads();
}
