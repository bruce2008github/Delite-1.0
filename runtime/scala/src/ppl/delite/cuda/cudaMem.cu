#include <stdio.h>
#include "hip/hip_runtime.h"
#include "ppl_delite_cuda_DeliteCudaDriver.h"

JNIEXPORT jlong JNICALL Java_ppl_delite_cuda_DeliteCudaDriver_cudaMemAlloc(JNIEnv *env, jobject obj, jint size)
{

	hipDeviceptr_t dptr;

	if(hipMalloc(&dptr, size) != hipSuccess)
	{
		printf("Error while allocating device memory of size %d\n", size);
		return -1;
	}
	
	return (jlong)dptr;
}

JNIEXPORT void JNICALL Java_ppl_delite_cuda_DeliteCudaDriver_cudaMemFree(JNIEnv *env, jobject obj, jlong devPtr)
{
	if(hipFree((hipDeviceptr_t)devPtr) != hipSuccess)
	{
		printf("ERROR: hipFree\n");
		return;
	}
}

JNIEXPORT jlong JNICALL Java_ppl_delite_cuda_DeliteCudaDriver_cudaMemAllocHost(JNIEnv *env, jobject obj, jint size)
{
	void *host_mem;
	//hipMemAllocHost(&host_mem, size);
	if(hipHostAlloc(&host_mem, size, hipHostMallocMapped) != hipSuccess)
	{
		printf("ERROR: hipHostAlloc\n");
		return -1;
	}

	return (jlong)host_mem;
}

JNIEXPORT jlong JNICALL Java_ppl_delite_cuda_DeliteCudaDriver_cudaMemHostGetDevicePointer(JNIEnv *env, jobject obj, jlong hostptr)
{
	hipDeviceptr_t dptr;

	if(hipHostGetDevicePointer(&dptr, (void *)hostptr, 0) != hipSuccess)
	{
		printf("ERROR: hipHostGetDevicePointer\n");
		return -1;
	}

	return (jlong)dptr;
}

/*
JNIEXPORT void JNICALL Java_ppl_delite_cuda_DeliteCudaDriver_cudaMemCpyHtoD(JNIEnv *env, jobject obj, jlong devPtr, jdoubleArray host_arr, jint offset, jint count)
{
	hipDeviceptr_t dptr = (hipDeviceptr_t)devPtr;

	jdouble *h_A = (jdouble *)malloc(count*sizeof(jdouble));

	env->GetDoubleArrayRegion(host_arr, offset, count, h_A);
	
	if(hipMemcpyHtoD(dptr, h_A, count*sizeof(jdouble)) != hipSuccess)
	{
		printf("ERROR: hipMemcpyHtoD\n");
		return;
	}

	free(h_A);
}


JNIEXPORT void JNICALL Java_ppl_delite_cuda_DeliteCudaDriver_cudaMemCpyDtoH(JNIEnv *env, jobject obj, jdoubleArray host_arr, jlong dev_ptr, jint offset, jint count)
{
	
	jdouble *h_A = (jdouble *)malloc(count*sizeof(jdouble));

	if(hipMemcpyDtoH(h_A, (hipDeviceptr_t)dev_ptr, count*sizeof(jdouble)) != hipSuccess)
	{
		printf("ERROR: hipMemcpyDtoH\n");
		return;
	}

	env->SetDoubleArrayRegion(host_arr, offset, count, h_A);

	free(h_A);
}


JNIEXPORT void JNICALL Java_ppl_delite_cuda_DeliteCudaDriver_cudaMemCpyDtoHNative(JNIEnv *env, jobject obj, jlong host_ptr, jlong dev_ptr, jint offset, jint count)
{
	if(hipMemcpyDtoH((void*)host_ptr, (hipDeviceptr_t)dev_ptr, count*sizeof(jdouble)) != hipSuccess)
	{
		printf("ERROR: cuMemcpyDtoHNative\n");
		return;
	}
}

JNIEXPORT void JNICALL Java_ppl_delite_cuda_DeliteCudaDriver_cudaMemCpyHtoDNative(JNIEnv *env, jobject obj, jlong dev_ptr, jlong host_ptr, jint offset, jint count)
{
	if(hipMemcpyHtoD((hipDeviceptr_t)dev_ptr, (void*)host_ptr, count*sizeof(jdouble)) != hipSuccess)
	{
		printf("ERROR: cuMemcpyDtoHNative\n");
		return;
	}
}
*/

JNIEXPORT void JNICALL Java_ppl_delite_cuda_DeliteCudaDriver_cudaMemCpyHtoDAsync(JNIEnv *env, jobject obj, jlong devPtr, jdoubleArray host_arr, jint offset, jint count, jlong h_A, jlong stream)
{
	hipDeviceptr_t dptr = (hipDeviceptr_t)devPtr;

	env->GetDoubleArrayRegion(host_arr, offset, count, (jdouble*)h_A);

	if(hipMemcpyHtoDAsync(dptr, (jdouble*)h_A, count*sizeof(jdouble), (hipStream_t)stream) != hipSuccess)
	{
		printf("ERROR: hipMemcpyHtoDAsync\n");
		return;
	}
	//hipStreamSynchronize((hipStream_t)stream);
}

JNIEXPORT void JNICALL Java_ppl_delite_cuda_DeliteCudaDriver_cudaMemCpyDtoHAsync(JNIEnv *env, jobject obj, jdoubleArray host_arr, jlong dev_ptr, jint offset, jint count, jlong h_A, jlong stream)
{
	if(hipMemcpyDtoHAsync((jdouble*)h_A, (hipDeviceptr_t)dev_ptr, count*sizeof(jdouble), (hipStream_t)stream) != hipSuccess)
	{
		printf("ERROR: hipMemcpyDtoHAsync\n");
		return;
	}

	hipStreamSynchronize((hipStream_t)stream);

	env->SetDoubleArrayRegion(host_arr, offset, count, (jdouble*)h_A);
}

JNIEXPORT void JNICALL Java_ppl_delite_cuda_DeliteCudaDriver_cudaMemCpyHtoDAsyncFloat(JNIEnv *env, jobject obj, jlong devPtr, jfloatArray host_arr, jint offset, jint count, jlong h_A, jlong stream)
{
	hipDeviceptr_t dptr = (hipDeviceptr_t)devPtr;

	env->GetFloatArrayRegion(host_arr, offset, count, (jfloat*)h_A);
	
	if(hipMemcpyHtoDAsync(dptr, (jfloat*)h_A, count*sizeof(jfloat), (hipStream_t)stream) != hipSuccess)
	{
		printf("ERROR: hipMemcpyHtoDAsync Float\n");
		return;
	}
	//hipStreamSynchronize((hipStream_t)stream);
}

JNIEXPORT void JNICALL Java_ppl_delite_cuda_DeliteCudaDriver_cudaMemCpyDtoHAsyncFloat(JNIEnv *env, jobject obj, jfloatArray host_arr, jlong dev_ptr, jint offset, jint count, jlong h_A, jlong stream)
{
	if(hipMemcpyDtoHAsync((jfloat*)h_A, (hipDeviceptr_t)dev_ptr, count*sizeof(jfloat), (hipStream_t)stream) != hipSuccess)
	{
		printf("ERROR: hipMemcpyDtoHAsync Float\n");
		return;
	}

	hipStreamSynchronize((hipStream_t)stream);

	env->SetFloatArrayRegion(host_arr, offset, count, (jfloat*)h_A);
}

JNIEXPORT void JNICALL Java_ppl_delite_cuda_DeliteCudaDriver_cudaMemCpyHtoDAsyncInt(JNIEnv *env, jobject obj, jlong devPtr, jintArray host_arr, jint offset, jint count, jlong h_A, jlong stream)
{
	hipDeviceptr_t dptr = (hipDeviceptr_t)devPtr;

	env->GetIntArrayRegion(host_arr, offset, count, (jint*)h_A);

	if(hipMemcpyHtoDAsync(dptr, (jint*)h_A, count*sizeof(jint), (hipStream_t)stream) != hipSuccess)
	{
		printf("ERROR: hipMemcpyHtoDAsync Int\n");
		return;
	}
	//hipStreamSynchronize((hipStream_t)stream);
}

JNIEXPORT void JNICALL Java_ppl_delite_cuda_DeliteCudaDriver_cudaMemCpyDtoHAsyncInt(JNIEnv *env, jobject obj, jintArray host_arr, jlong dev_ptr, jint offset, jint count, jlong h_A, jlong stream)
{
	if(hipMemcpyDtoHAsync((jint*)h_A, (hipDeviceptr_t)dev_ptr, count*sizeof(jint), (hipStream_t)stream) != hipSuccess)
	{
		printf("ERROR: hipMemcpyDtoHAsync Int\n");
		return;
	}

	hipStreamSynchronize((hipStream_t)stream);

	env->SetIntArrayRegion(host_arr, offset, count, (jint*)h_A);
}


JNIEXPORT void JNICALL Java_ppl_delite_cuda_DeliteCudaDriver_cudaMemCpyDtoD(JNIEnv *env, jobject obj, jlong dst_ptr, jlong src_ptr, jint size)
{
	if(hipMemcpyDtoD((hipDeviceptr_t)dst_ptr, (hipDeviceptr_t)src_ptr, size) != hipSuccess)
	{
		printf("ERROR: hipMemcpyDtoD\n");
		return;
	}
}
