#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2009 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and 
 * proprietary rights in and to this software and related documentation and 
 * any modifications thereto.  Any use, reproduction, disclosure, or distribution 
 * of this software and related documentation without an express license 
 * agreement from NVIDIA Corporation is strictly prohibited.
 * 
 */

/* Matrix multiplication: C = A * B.
 * Device code.
 */

#include "matrixMul.h"
#define AS(i, j) As[i][j]
#define BS(i, j) Bs[i][j]

////////////////////////////////////////////////////////////////////////////////
//! Matrix multiplication on the device: C = A * B
//! wA is A's width and wB is B's width
////////////////////////////////////////////////////////////////////////////////
__global__ void matrixMul( double* A, double* B, double* C, int wA, int wB, int hA)
{
    // Block index
    int bx = blockIdx.x;
    int by = blockIdx.y;

    // Thread index
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    // Index of the first sub-matrix of A processed by the block
    int aBegin = wA * BLOCK_SIZE * by;

    // Index of the last sub-matrix of A processed by the block
    int aEnd   = aBegin + wA - 1;

    // Step size used to iterate through the sub-matrices of A
    int aStep  = BLOCK_SIZE;

    // Index of the first sub-matrix of B processed by the block
    int bBegin = BLOCK_SIZE * bx;

    // Step size used to iterate through the sub-matrices of B
    int bStep  = BLOCK_SIZE * wB;

    // Csub is used to store the element of the block sub-matrix
    // that is computed by the thread
    double Csub = 0;

	int iter = 0;
    // Loop over all the sub-matrices of A and B
    // required to compute the block sub-matrix
    for (int a = aBegin, b = bBegin;
             a <= aEnd;
             a += aStep, b += bStep) {
	
        // Declaration of the shared memory array As used to
        // store the sub-matrix of A
        __shared__ double As[BLOCK_SIZE][BLOCK_SIZE];

        // Declaration of the shared memory array Bs used to
        // store the sub-matrix of B
        __shared__ double Bs[BLOCK_SIZE][BLOCK_SIZE];

        // Load the matrices from device memory
        // to shared memory; each thread loads
        // one element of each matrix
		AS(ty, tx) = 0;
		BS(ty, tx) = 0;
		if((iter*BLOCK_SIZE+tx<wA) && (by*BLOCK_SIZE+ty<hA))
        	AS(ty, tx) = A[a + wA * ty + tx];
		if((iter*BLOCK_SIZE+ty<wA) && (bx*BLOCK_SIZE+tx<wB))
        	BS(ty, tx) = B[b + wB * ty + tx];

        // Synchronize to make sure the matrices are loaded
        __syncthreads();

        // Multiply the two matrices together;
        // each thread computes one element
        // of the block sub-matrix
        for (int k = 0; k < BLOCK_SIZE; ++k)
            Csub += AS(ty, k) * BS(k, tx);

        // Synchronize to make sure that the preceding
        // computation is done before loading two new
        // sub-matrices of A and B in the next iteration
        __syncthreads();
		iter += 1;
    }

    // Write the block sub-matrix to device memory;
    // each thread writes one element
    int c = wB * BLOCK_SIZE * by + BLOCK_SIZE * bx;

	if((bx*BLOCK_SIZE+tx<wB) && (by*BLOCK_SIZE+ty<hA))
    	C[c + wB * ty + tx] = Csub;
}

