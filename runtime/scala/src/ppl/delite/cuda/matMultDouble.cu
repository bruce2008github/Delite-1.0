
#include <hip/hip_runtime.h>
__device__ void rank1_updateD( double a, const double *b, double *c )
{
	c[0] += a*b[0];
	c[1] += a*b[1];
	c[2] += a*b[2];
	c[3] += a*b[3];
	c[4] += a*b[4];
	c[5] += a*b[5];
	c[6] += a*b[6];
	c[7] += a*b[7];
	c[8] += a*b[8];
	c[9] += a*b[9];
	c[10] += a*b[10];
	c[11] += a*b[11];
	c[12] += a*b[12];
	c[13] += a*b[13];
	c[14] += a*b[14];
	c[15] += a*b[15];
}

__device__ void rankk_updateD( int k, const double *A, int lda, const double *b, int ldb, double *c )
{
    if( k <= 0 ) return;

    int i = 0;
    rank1_updateD( A[0], &b[i*ldb], c ); if( ++i >= k ) return; A += lda;
    rank1_updateD( A[0], &b[i*ldb], c ); if( ++i >= k ) return; A += lda;
    rank1_updateD( A[0], &b[i*ldb], c ); if( ++i >= k ) return; A += lda;
    rank1_updateD( A[0], &b[i*ldb], c ); if( ++i >= k ) return; A += lda;
    
    rank1_updateD( A[0], &b[i*ldb], c ); if( ++i >= k ) return; A += lda;
    rank1_updateD( A[0], &b[i*ldb], c ); if( ++i >= k ) return; A += lda;
    rank1_updateD( A[0], &b[i*ldb], c ); if( ++i >= k ) return; A += lda;
    rank1_updateD( A[0], &b[i*ldb], c ); if( ++i >= k ) return; A += lda;
    
    rank1_updateD( A[0], &b[i*ldb], c ); if( ++i >= k ) return; A += lda;
    rank1_updateD( A[0], &b[i*ldb], c ); if( ++i >= k ) return; A += lda;
    rank1_updateD( A[0], &b[i*ldb], c ); if( ++i >= k ) return; A += lda;
    rank1_updateD( A[0], &b[i*ldb], c ); if( ++i >= k ) return; A += lda;
    
    rank1_updateD( A[0], &b[i*ldb], c ); if( ++i >= k ) return; A += lda;
    rank1_updateD( A[0], &b[i*ldb], c ); if( ++i >= k ) return; A += lda;
    rank1_updateD( A[0], &b[i*ldb], c );
}

__device__ void store_blockD( int num, double alpha, double *c, double beta, double *C, int ldc )
{
    if( num <= 0 ) return;

    if( beta == 0 )
    {
        //
        //  for the case when C is initialized with inf or NaN
        //
        int i = 0; 
        C[0] = alpha*c[i++]; if( i >= num ) return; C += ldc;  
        C[0] = alpha*c[i++]; if( i >= num ) return; C += ldc;  
        C[0] = alpha*c[i++]; if( i >= num ) return; C += ldc;  
        C[0] = alpha*c[i++]; if( i >= num ) return; C += ldc;  
        
        C[0] = alpha*c[i++]; if( i >= num ) return; C += ldc;  
        C[0] = alpha*c[i++]; if( i >= num ) return; C += ldc;  
        C[0] = alpha*c[i++]; if( i >= num ) return; C += ldc;  
        C[0] = alpha*c[i++]; if( i >= num ) return; C += ldc;  

        C[0] = alpha*c[i++]; if( i >= num ) return; C += ldc;  
        C[0] = alpha*c[i++]; if( i >= num ) return; C += ldc;  
        C[0] = alpha*c[i++]; if( i >= num ) return; C += ldc;  
        C[0] = alpha*c[i++]; if( i >= num ) return; C += ldc;  

        C[0] = alpha*c[i++]; if( i >= num ) return; C += ldc;  
        C[0] = alpha*c[i++]; if( i >= num ) return; C += ldc;  
        C[0] = alpha*c[i++]; if( i >= num ) return; C += ldc;  
        C[0] = alpha*c[i++];
    }
    else
    {
        int i = 0; 
        C[0] = alpha*c[i++] + beta*C[0]; if( i >= num ) return; C += ldc;  
        C[0] = alpha*c[i++] + beta*C[0]; if( i >= num ) return; C += ldc;  
        C[0] = alpha*c[i++] + beta*C[0]; if( i >= num ) return; C += ldc;  
        C[0] = alpha*c[i++] + beta*C[0]; if( i >= num ) return; C += ldc;  
        
        C[0] = alpha*c[i++] + beta*C[0]; if( i >= num ) return; C += ldc;  
        C[0] = alpha*c[i++] + beta*C[0]; if( i >= num ) return; C += ldc;  
        C[0] = alpha*c[i++] + beta*C[0]; if( i >= num ) return; C += ldc;  
        C[0] = alpha*c[i++] + beta*C[0]; if( i >= num ) return; C += ldc;  

        C[0] = alpha*c[i++] + beta*C[0]; if( i >= num ) return; C += ldc;  
        C[0] = alpha*c[i++] + beta*C[0]; if( i >= num ) return; C += ldc;  
        C[0] = alpha*c[i++] + beta*C[0]; if( i >= num ) return; C += ldc;  
        C[0] = alpha*c[i++] + beta*C[0]; if( i >= num ) return; C += ldc;  

        C[0] = alpha*c[i++] + beta*C[0]; if( i >= num ) return; C += ldc;  
        C[0] = alpha*c[i++] + beta*C[0]; if( i >= num ) return; C += ldc;  
        C[0] = alpha*c[i++] + beta*C[0]; if( i >= num ) return; C += ldc;  
        C[0] = alpha*c[i++] + beta*C[0];
    }
}

//
//  C = alpha*A*B + beta*C
//
/*
	lmem = 0
	smem = 1168
	reg  = 30
	active threads = 512 
 */
 
__global__ void   dgemmNN_device(double *A_ptr, double *B_ptr, double *C_ptr, int widthA, int widthB, int heightA) 
//__global__ void   dgemmNN_device( int m, int n, const double *A, int lda, 
//const double *B, int ldb, double* C, int ldc, int k, double alpha, double beta )
{
	int m = widthB;
	int n = heightA;
	double *A = B_ptr;
	int lda = widthB;
	double *B = A_ptr;
	int ldb = widthA;
	double *C = C_ptr;
	int ldc = widthB;
	int k = widthA;
	double alpha = 1.0;
	double beta = 0.0;

	const int inx = threadIdx.x;
	const int iny = threadIdx.y;
	const int ibx = blockIdx.x * 64;
	const int iby = blockIdx.y * 16;
	const int row = ibx + inx + iny*16;
	
	A += row;
	B += inx + ( iby + iny ) * ldb;
	C += row  + iby * ldc;
	
	double c[16] = {0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0};
    
	__shared__ double b[16][17];
	for( ; k > 0; k -= 16 )
	{
#pragma unroll
		for( int i = 0; i < 16; i += 4 )
			b[inx][iny+i]  = B[i*ldb];
		__syncthreads();

        if( k < 16 )  break;

#pragma unroll
	    for( int i = 0; i < 16; i++, A += lda )
		    rank1_updateD( A[0], &b[i][0], c ); 
	    __syncthreads();
		
		B += 16;
	};

    rankk_updateD( k, A, lda, &b[0][0], 17, c );

    if( row >= m )  return;
    
    store_blockD( n - iby, alpha, c, beta, C, ldc);
}	


